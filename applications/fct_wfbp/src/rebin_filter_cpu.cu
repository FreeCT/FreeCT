#include "hip/hip_runtime.h"
/* FreeCT_wFBP is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/

#define pi 3.1415926535897f

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <complex.h>
#include <fftw3.h>

#include <interp.h>
#include <recon_structs.h>
#include <rebin_filter_cpu.h>

void copy_sheet(float * sheetptr, int row, struct recon_metadata *mr, struct ct_geom cg);
void load_filter(float * f_array,struct recon_metadata * mr);

inline float angle(float x1,float x2,float y1,float y2){
    return asin((x1*y2-x2*y1)/(sqrt(x1*x1+x2*x2)*sqrt(y1*y1+y2*y2)));
}

inline float beta_rk(float da,float dr,float channel,int os_flag, struct ct_geom cg){
    float b0=(channel-pow(2.0f,os_flag)*cg.central_channel)*(cg.fan_angle_increment/pow(2.0f,os_flag));
    return angle(-(cg.r_f+dr),-(da),-(cg.src_to_det*cos(b0)+dr),-(cg.src_to_det*sin(b0)+da));
}

inline float d_alpha_r(float da,float dr,struct ct_geom cg){
    return angle(cg.r_f,0,cg.r_f+dr,da);
}

inline float r_fr(float da, float dr,struct ct_geom cg){
    return sqrt((cg.r_f+dr)*(cg.r_f+dr)+da*da);
}

inline float get_beta_idx(float beta,float * beta_lookup,int n_elements){
    int idx_low=0;

    while (beta>beta_lookup[idx_low]&&idx_low<(n_elements-1)){
    	idx_low++;
    }

    if (idx_low==0)
	idx_low++; 
    
    return (float)idx_low-1.0f+(beta-beta_lookup[idx_low-1])/(beta_lookup[idx_low]-beta_lookup[idx_low-1]);
}


void filter_cpu(float * row, float * filter, int N);

void rebin_nffs_cpu(struct recon_metadata *mr);
void rebin_pffs_cpu(struct recon_metadata *mr);
void rebin_zffs_cpu(struct recon_metadata *mr);
void rebin_affs_cpu(struct recon_metadata *mr);

int rebin_filter_cpu(struct recon_metadata * mr){

    switch (mr->ri.n_ffs){
    case 1:{
	rebin_nffs_cpu(mr);
	break;}
    case 2:{
	if (mr->rp.z_ffs==1)
	    rebin_zffs_cpu(mr);
	else
	    rebin_pffs_cpu(mr);
	break;}
    case 4:{
	rebin_affs_cpu(mr);
	break;}
    }
    
    return 0;
}

void rebin_nffs_cpu(struct recon_metadata *mr){
    const struct ct_geom cg=mr->cg;

    float * h_output=(float*)calloc(cg.n_channels_oversampled*cg.n_rows*mr->ri.n_proj_pull/mr->ri.n_ffs,sizeof(float));

    // Main loop
    int n_proj=mr->ri.n_proj_pull/mr->ri.n_ffs;
    struct array_dims d;
    d.idx1=cg.n_channels;
    d.idx2=cg.n_rows;
    d.idx3=n_proj;
    
    for (int channel=0;channel<cg.n_channels_oversampled;channel++){
	const float beta=asin(((float)channel-2*cg.central_channel)*(cg.fan_angle_increment/2));
	float beta_idx=beta/cg.fan_angle_increment+cg.central_channel;
	for (int proj=0;proj<n_proj;proj++){
	    float alpha_idx=(float)proj-beta*cg.n_proj_turn/(2.0f*pi);
	    for (int row=0;row<cg.n_rows;row++){
		int out_idx=cg.n_channels_oversampled*cg.n_rows*proj+cg.n_channels_oversampled*row+channel;
		h_output[out_idx]=interp3(mr->ctd.raw,d,beta_idx,row,alpha_idx);
	    }
	}
    }
    
    //Copy data into our mr structure, skipping initial truncated projections
    size_t offset=cg.add_projections;
    for (int i=0;i<cg.n_channels_oversampled;i++){
	for (int j=0;j<cg.n_rows;j++){
	    for (int k=0;k<(mr->ri.n_proj_pull/mr->ri.n_ffs-2*cg.add_projections);k++){

		int out_idx=k*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i;
		int in_idx=(k+offset)*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i;

		mr->ctd.rebin[out_idx]=h_output[in_idx];
	    }
	}
    }

    
    printf("Filtering...\n");
    
    // Load and run filter
    float * h_filter=(float*)calloc(2*cg.n_channels_oversampled,sizeof(float));
    load_filter(h_filter,mr);

    for (int i=0;i<(n_proj-2*cg.add_projections);i++){
	for (int j=0;j<cg.n_rows;j++){
	    int row_start_idx=i*cg.n_channels_oversampled*cg.n_rows+cg.n_channels_oversampled*j;
	    filter_cpu(&mr->ctd.rebin[row_start_idx],h_filter,cg.n_channels_oversampled);
	}
    }
    
    // Check "testing" flag, write rebin to disk if set
    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/rebin.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->ctd.rebin,sizeof(float),cg.n_channels_oversampled*cg.n_rows*(mr->ri.n_proj_pull-2*cg.add_projections_ffs)/mr->ri.n_ffs,outfile);
	fclose(outfile);
    }

    free(h_output);
    free(h_filter);
}

void rebin_pffs_cpu(struct recon_metadata *mr){
    // Set up some constants
    struct ct_geom cg=mr->cg;
    struct recon_info ri=mr->ri;
    const double da=cg.src_to_det*cg.r_f*cg.fan_angle_increment/(4.0f*(cg.src_to_det-cg.r_f));
    int n_proj=mr->ri.n_proj_pull/mr->ri.n_ffs;

    // Allocate raw data arrays and intermediate output array
    float * raw_1;
    raw_1=(float*)malloc(cg.n_channels*cg.n_rows*n_proj*sizeof(float));
    float * raw_2;
    raw_2=(float*)malloc(cg.n_channels*cg.n_rows*n_proj*sizeof(float));
    
    float * rebin_t;
    rebin_t=(float*)malloc(cg.n_channels_oversampled*cg.n_rows*n_proj*sizeof(float));

    // Split raw data by focal spot
    for (int i=0;i<n_proj;i++){
	for (int j=0;j<cg.n_rows;j++){
	    for (int k=0;k<cg.n_channels;k++){
		int out_idx=cg.n_channels*cg.n_rows*i+cg.n_channels*j+k;
		int in_idx_ffs1=cg.n_channels*cg.n_rows*(2*i)+cg.n_channels*j+k;
		int in_idx_ffs2=cg.n_channels*cg.n_rows*(2*i+1)+cg.n_channels*j+k;

		raw_1[out_idx]=mr->ctd.raw[in_idx_ffs1];
		raw_2[out_idx]=mr->ctd.raw[in_idx_ffs2];
	    }
	}
    }

    struct array_dims dim;
    dim.idx1=cg.n_channels;
    dim.idx2=cg.n_rows;
    dim.idx3=n_proj;

    float * beta_lookup;
    beta_lookup=(float*)malloc(cg.n_channels_oversampled*sizeof(float));
    
    // Rebin over angles
    for (int proj=0;proj<n_proj;proj++){
	for (int row=0;row<cg.n_rows;row++){
	    for (int channel=0;channel<cg.n_channels;channel++){
		
		int out_idx_1=proj*cg.n_channels_oversampled*cg.n_rows+row*cg.n_channels_oversampled+2*channel;
		int out_idx_2=proj*cg.n_channels_oversampled*cg.n_rows+row*cg.n_channels_oversampled+2*channel+1;

		// +da
		float beta_1= beta_rk(da,0,channel,0,cg);
		beta_lookup[2*channel]=beta_1;
		float alpha_idx_1=ri.n_ffs*(proj)-beta_1*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(da,0,cg)*cg.n_proj_ffs/(2.0f*pi);
		
		// -da
		float beta_2 = beta_rk(-da,0,channel,0,cg);
		beta_lookup[2*channel+1]=beta_2;
		float alpha_idx_2=ri.n_ffs*(proj)-beta_2*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(-da,0,cg)*cg.n_proj_ffs/(2.0f*pi);

		// Rescale alpha indices to properly index the raw arrays as 0, 1, 2, 3, ...
		alpha_idx_1=alpha_idx_1/2.0f; // raw_1 contains alpha projections 0, 2, 4, 6, ...
		alpha_idx_2=(alpha_idx_2-1.0f)/2.0f; // raw_2 contains projections 1, 3, 5, 7, ...
		
		rebin_t[out_idx_1]=interp3(raw_1,dim,channel,row,alpha_idx_1);
		rebin_t[out_idx_2]=interp3(raw_2,dim,channel,row,alpha_idx_2);
	    }
	}
    }
    
    // Free any arrays we no longer need, allocate final output array
    free(raw_1);
    free(raw_2);
    
    float * h_output;
    h_output=(float*)malloc(cg.n_channels_oversampled*cg.n_rows*n_proj*sizeof(float));

    // Update the interpolation array dimensions since new array has twice as many channels
    dim.idx1*=2;
    
    // Rebin channels
    for (int proj=0;proj<n_proj;proj++){
	for (int row=0;row<cg.n_rows;row++){
	    for (int channel=0;channel<cg.n_channels_oversampled;channel++){
		float beta  = asin((channel-2*cg.central_channel)*(cg.fan_angle_increment/2));
		//float beta_idx=beta/(cg.fan_angle_increment/2.0f)+2.0f*cg.central_channel;
		float beta_idx=get_beta_idx(beta,beta_lookup,cg.n_channels_oversampled);
		int out_idx=cg.n_channels_oversampled*cg.n_rows*proj+cg.n_channels_oversampled*row+channel;
		h_output[out_idx]=interp3(rebin_t,dim,beta_idx,row,proj);
	    }
	}
    }

    free(rebin_t);
    
    //Copy data into our mr structure, skipping initial truncated projections
    size_t offset=cg.add_projections;
    for (int i=0;i<cg.n_channels_oversampled;i++){
	for (int j=0;j<cg.n_rows;j++){
	    for (int k=0;k<(mr->ri.n_proj_pull/mr->ri.n_ffs-2*cg.add_projections);k++){
		mr->ctd.rebin[k*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i]=h_output[(k+offset)*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i];
	    }
	}
    }

    printf("Filtering...\n");
    
    // Load and run filter
    float * h_filter=(float*)calloc(2*cg.n_channels_oversampled,sizeof(float));
    load_filter(h_filter,mr);

    for (int i=0;i<(n_proj-2*cg.add_projections);i++){
	for (int j=0;j<cg.n_rows;j++){
	    int row_start_idx=i*cg.n_channels_oversampled*cg.n_rows+cg.n_channels_oversampled*j;
	    filter_cpu(&mr->ctd.rebin[row_start_idx],h_filter,cg.n_channels_oversampled);
	}
    }
    
    // Check "testing" flag, write rebin to disk if set
    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/rebin.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->ctd.rebin,sizeof(float),cg.n_channels_oversampled*cg.n_rows*(mr->ri.n_proj_pull-2*cg.add_projections_ffs)/mr->ri.n_ffs,outfile);
	fclose(outfile);
    }

    free(h_output);
    free(h_filter);

}

void rebin_zffs_cpu(struct recon_metadata *mr){

    // Set up some constants
    struct ct_geom cg=mr->cg;
    struct recon_info ri=mr->ri;
    struct recon_params rp=mr->rp;

    const double da=0.0;
    const double dr=cg.src_to_det*rp.coll_slicewidth/(4.0*(cg.src_to_det-cg.r_f)*tan(cg.anode_angle));
    int n_proj=mr->ri.n_proj_pull/mr->ri.n_ffs;
    
    // Allocate raw data arrays and final output array
    float * raw_1;
    raw_1=(float*)malloc(cg.n_channels*cg.n_rows*n_proj*sizeof(float));
    float * raw_2;
    raw_2=(float*)malloc(cg.n_channels*cg.n_rows*n_proj*sizeof(float));
    
    float * h_output;
    h_output=(float*)malloc(cg.n_channels_oversampled*cg.n_rows*n_proj*sizeof(float));

    // Split raw data by focal spot
    for (int i=0;i<n_proj;i++){
	for (int j=0;j<cg.n_rows_raw;j++){
	    for (int k=0;k<cg.n_channels;k++){
		int out_idx=cg.n_channels*cg.n_rows_raw*i+cg.n_channels*j+k;
		int in_idx_ffs1=cg.n_channels*cg.n_rows_raw*(2*i)+cg.n_channels*j+k;
		int in_idx_ffs2=cg.n_channels*cg.n_rows_raw*(2*i+1)+cg.n_channels*j+k;

		raw_1[out_idx]=mr->ctd.raw[in_idx_ffs1];
		raw_2[out_idx]=mr->ctd.raw[in_idx_ffs2];
	    }
	}
    }

    // Allocate and compute beta lookup tables
    float * beta_lookup_1;
    float * beta_lookup_2;
    beta_lookup_1=(float*)malloc(cg.n_channels*sizeof(float));
    beta_lookup_2=(float*)malloc(cg.n_channels*sizeof(float));
    for (int i=0;i<cg.n_channels;i++){
	beta_lookup_1[i]=beta_rk(da,-dr,i,0,cg);
	beta_lookup_2[i]=beta_rk(da, dr,i,0,cg);
    }

    // Set up interpolation array dims
    struct array_dims dim;
    dim.idx1=cg.n_channels;
    dim.idx2=cg.n_rows_raw;
    dim.idx3=n_proj;

    for (int proj=0;proj<n_proj;proj++){
	for (int row=0;row<cg.n_rows_raw;row++){
	    for (int channel=0;channel<cg.n_channels_oversampled;channel++){

		// da=0, dr= -dr
 		float beta_1=asin((channel-2.0f*cg.central_channel)*(cg.fan_angle_increment/2.0f)*cg.r_f/r_fr(0.0f,-dr,cg));
		float alpha_idx_1=ri.n_ffs*(proj)-beta_1*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(da,-dr,cg)*cg.n_proj_ffs/(2.0f*pi);
		float beta_idx_1=get_beta_idx(beta_1,beta_lookup_1,cg.n_channels);

		// da=0, dr= +dr
		float beta_2=asin((channel-2.0f*cg.central_channel)*(cg.fan_angle_increment/2.0f)*cg.r_f/r_fr(0.0f,dr,cg));
		float alpha_idx_2=ri.n_ffs*(proj)-beta_2*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(da,dr,cg)*cg.n_proj_ffs/(2.0f*pi);
		float beta_idx_2=get_beta_idx(beta_2,beta_lookup_2,cg.n_channels);

		// Rescale alpha indices to properly index the raw arrays as 0, 1, 2, 3, ...
		alpha_idx_1=alpha_idx_1/2.0f; // raw_1 contains alpha projections 0, 2, 4, 6, ...
		alpha_idx_2=(alpha_idx_2-1.0f)/2.0f; // raw_2 contains projections 1, 3, 5, 7, ...

		int out_idx_1;
		int out_idx_2;
		
		if (!cg.reverse_row_interleave){
		    out_idx_1=cg.n_channels_oversampled*cg.n_rows*proj + cg.n_channels_oversampled*  2*row   + channel;
		    out_idx_2=cg.n_channels_oversampled*cg.n_rows*proj + cg.n_channels_oversampled*(2*row+1) + channel;
		}
		else{
		    out_idx_1=cg.n_channels_oversampled*cg.n_rows*proj + cg.n_channels_oversampled* (2*row+1) + channel;
		    out_idx_2=cg.n_channels_oversampled*cg.n_rows*proj + cg.n_channels_oversampled*   2*row   + channel;
		}

		h_output[out_idx_1]=interp3(raw_1,dim,beta_idx_1,row,alpha_idx_1);
		h_output[out_idx_2]=interp3(raw_2,dim,beta_idx_2,row,alpha_idx_2);
	    }
	}
    }

    //Copy data into our mr structure, skipping initial truncated projections
    size_t offset=cg.add_projections;
    for (int i=0;i<cg.n_channels_oversampled;i++){
	for (int j=0;j<cg.n_rows;j++){
	    for (int k=0;k<(mr->ri.n_proj_pull/mr->ri.n_ffs-2*cg.add_projections);k++){
		mr->ctd.rebin[k*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i]=h_output[(k+offset)*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i];
	    }
	}
    }

    printf("Filtering...\n");
    
    // Load and run filter
    float * h_filter=(float*)calloc(2*cg.n_channels_oversampled,sizeof(float));
    load_filter(h_filter,mr);

    for (int i=0;i<(n_proj-2*cg.add_projections);i++){
	for (int j=0;j<cg.n_rows;j++){
	    int row_start_idx=i*cg.n_channels_oversampled*cg.n_rows+cg.n_channels_oversampled*j;
	    filter_cpu(&mr->ctd.rebin[row_start_idx],h_filter,cg.n_channels_oversampled);
	}
    }
    
    // Check "testing" flag, write rebin to disk if set
    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/rebin.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->ctd.rebin,sizeof(float),cg.n_channels_oversampled*cg.n_rows*(mr->ri.n_proj_pull-2*cg.add_projections_ffs)/mr->ri.n_ffs,outfile);
	fclose(outfile);
    }

    free(raw_1);
    free(raw_2);
    free(beta_lookup_1);
    free(beta_lookup_2);
    free(h_output);
    free(h_filter);

}

void rebin_affs_cpu(struct recon_metadata *mr){
    // Set up some constants
    struct ct_geom cg=mr->cg;
    struct recon_info ri=mr->ri;
    struct recon_params rp=mr->rp;

    const double da=cg.src_to_det*cg.r_f*cg.fan_angle_increment/(4.0f*(cg.src_to_det-cg.r_f));
    const double dr=cg.src_to_det*rp.coll_slicewidth/(4.0*(cg.src_to_det-cg.r_f)*tan(cg.anode_angle));
    int n_proj=mr->ri.n_proj_pull/mr->ri.n_ffs;

    // Allocate raw data arrays and final output array
    float * raw_1;
    raw_1=(float*)malloc(cg.n_channels*cg.n_rows_raw*n_proj*sizeof(float));
    float * raw_2;
    raw_2=(float*)malloc(cg.n_channels*cg.n_rows_raw*n_proj*sizeof(float));
    float * raw_3;
    raw_3=(float*)malloc(cg.n_channels*cg.n_rows_raw*n_proj*sizeof(float));
    float * raw_4;
    raw_4=(float*)malloc(cg.n_channels*cg.n_rows_raw*n_proj*sizeof(float));

    float * rebin_t_1;
    rebin_t_1=(float*)malloc(cg.n_channels_oversampled*cg.n_rows_raw*n_proj*sizeof(float));
    float * rebin_t_2;
    rebin_t_2=(float*)malloc(cg.n_channels_oversampled*cg.n_rows_raw*n_proj*sizeof(float));
    
    float * h_output;
    h_output=(float*)malloc(cg.n_channels_oversampled*cg.n_rows*n_proj*sizeof(float));

    // Split raw data by focal spot
    for (int i=0;i<n_proj;i++){
	for (int j=0;j<cg.n_rows_raw;j++){
	    for (int k=0;k<cg.n_channels;k++){
		int out_idx=cg.n_channels*cg.n_rows_raw*i+cg.n_channels*j+k;
		int in_idx_ffs1=cg.n_channels*cg.n_rows_raw*(4*i)+cg.n_channels*j+k;
		int in_idx_ffs2=cg.n_channels*cg.n_rows_raw*(4*i+1)+cg.n_channels*j+k;
		int in_idx_ffs3=cg.n_channels*cg.n_rows_raw*(4*i+2)+cg.n_channels*j+k;
		int in_idx_ffs4=cg.n_channels*cg.n_rows_raw*(4*i+3)+cg.n_channels*j+k;
		raw_1[out_idx]=mr->ctd.raw[in_idx_ffs1];
		raw_2[out_idx]=mr->ctd.raw[in_idx_ffs2];
		raw_3[out_idx]=mr->ctd.raw[in_idx_ffs3];
		raw_4[out_idx]=mr->ctd.raw[in_idx_ffs4];
	    }
	}
    }

    // Check 'testing' flag, and write to disk if set
    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/reshape_1.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(raw_1,sizeof(float),cg.n_channels*cg.n_rows_raw*ri.n_proj_pull/ri.n_ffs,outfile);
	fclose(outfile);

	memset(fullpath,0,4096+255);
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/reshape_2.ct_test");
	outfile=fopen(fullpath,"w");
	fwrite(raw_2,sizeof(float),cg.n_channels*cg.n_rows_raw*ri.n_proj_pull/ri.n_ffs,outfile);
	fclose(outfile);

	memset(fullpath,0,4096+255);
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/reshape_3.ct_test");
	outfile=fopen(fullpath,"w");
	fwrite(raw_3,sizeof(float),cg.n_channels*cg.n_rows_raw*ri.n_proj_pull/ri.n_ffs,outfile);
	fclose(outfile);

	memset(fullpath,0,4096+255);
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/reshape_4.ct_test");
	outfile=fopen(fullpath,"w");
	fwrite(raw_4,sizeof(float),cg.n_channels*cg.n_rows_raw*ri.n_proj_pull/ri.n_ffs,outfile);
	fclose(outfile);
    }
    
    struct array_dims dim;
    dim.idx1=cg.n_channels;
    dim.idx2=cg.n_rows_raw;
    dim.idx3=n_proj;

    // Allocate beta lookup tables
    // Allocate and compute beta lookup tables
    float * beta_lookup_1;
    float * beta_lookup_2;
    beta_lookup_1=(float*)malloc(sizeof(float)*cg.n_channels_oversampled);
    beta_lookup_2=(float*)malloc(sizeof(float)*cg.n_channels_oversampled);

    // Rebin projections
    for (int proj=0;proj<n_proj;proj++){
	for (int row=0;row<cg.n_rows_raw;row++){
	    for (int channel=0;channel<cg.n_channels;channel++){
		
		int out_idx_1=proj*cg.n_channels_oversampled*cg.n_rows_raw+row*cg.n_channels_oversampled+2*channel;
		int out_idx_2=proj*cg.n_channels_oversampled*cg.n_rows_raw+row*cg.n_channels_oversampled+2*channel+1;
		int out_idx_3=proj*cg.n_channels_oversampled*cg.n_rows_raw+row*cg.n_channels_oversampled+2*channel;
		int out_idx_4=proj*cg.n_channels_oversampled*cg.n_rows_raw+row*cg.n_channels_oversampled+2*channel+1;

		// -dr >>>>>
		// +da
		float beta_1 = beta_rk(da,-dr,channel,0,cg);
		float alpha_idx_1=ri.n_ffs*(proj)-beta_1*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(da,-dr,cg)*cg.n_proj_ffs/(2.0f*pi);
		// -da
		float beta_2 = beta_rk(-da,-dr,channel,0,cg);
		float alpha_idx_2=ri.n_ffs*(proj)-beta_2*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(-da,-dr,cg)*cg.n_proj_ffs/(2.0f*pi);
		beta_lookup_1[2*channel]=beta_1;
		beta_lookup_1[2*channel+1]=beta_2;		
		// <<<<< -dr

		// +dr >>>>>
		// +da
		float beta_3 = beta_rk(da,dr,channel,0,cg);
		//float alpha_idx_3=ri.n_ffs*(proj)-beta_1*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(da,dr,cg)*cg.n_proj_ffs/(2.0f*pi);
		float alpha_idx_3=ri.n_ffs*(proj)-beta_3*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(da,dr,cg)*cg.n_proj_ffs/(2.0f*pi);		
		// -da
		float beta_4 = beta_rk(-da,dr,channel,0,cg);
		//float alpha_idx_4=ri.n_ffs*(proj)-beta_2*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(-da,dr,cg)*cg.n_proj_ffs/(2.0f*pi);
		float alpha_idx_4=ri.n_ffs*(proj)-beta_4*cg.n_proj_ffs/(2.0f*pi)-d_alpha_r(-da,dr,cg)*cg.n_proj_ffs/(2.0f*pi);		
		beta_lookup_2[2*channel]=beta_3;
		beta_lookup_2[2*channel+1]=beta_4;
		// <<<<< +dr

		// Rescale alpha indices to properly index the raw arrays as 0, 1, 2, 3, ...
		alpha_idx_1=    alpha_idx_1    /4.0f; // raw_1 contains projections 0, 4, 8, 12, ...
		alpha_idx_2=(alpha_idx_2-1.0f) /4.0f; // raw_2 contains projections 1, 5, 9, 13, ...
		alpha_idx_3=(alpha_idx_3-2.0f) /4.0f; // raw_3 contains projections 2, 6, 10, 14, ...
		alpha_idx_4=(alpha_idx_4-3.0f) /4.0f; // raw_4 contains projections 3, 7, 11, 15, ...
		
		rebin_t_1[out_idx_1]=interp3(raw_1,dim,channel,row,alpha_idx_1);
		rebin_t_1[out_idx_2]=interp3(raw_2,dim,channel,row,alpha_idx_2);
		rebin_t_2[out_idx_3]=interp3(raw_3,dim,channel,row,alpha_idx_3);
		rebin_t_2[out_idx_4]=interp3(raw_4,dim,channel,row,alpha_idx_4);
	    }
	}
    }

    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/rebin_t1.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(rebin_t_1,sizeof(float),cg.n_channels_oversampled*cg.n_rows_raw*ri.n_proj_pull/ri.n_ffs,outfile);
	fclose(outfile);
	
	memset(fullpath,0,4096+255);
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/rebin_t2.ct_test");
	outfile=fopen(fullpath,"w");
	fwrite(rebin_t_2,sizeof(float),cg.n_channels_oversampled*cg.n_rows_raw*ri.n_proj_pull/ri.n_ffs,outfile);
	fclose(outfile);
    }

    free(raw_1);
    free(raw_2);
    free(raw_3);
    free(raw_4);

    // Update the interpolation array dimensions since new array has twice as many channels
    dim.idx1*=2;

    // Rebin channels
    for (int proj=0;proj<n_proj;proj++){
	for (int row=0;row<cg.n_rows_raw;row++){
	    for (int channel=0;channel<cg.n_channels_oversampled;channel++){

		int out_idx_1;
		int out_idx_2;
		
		if (!cg.reverse_row_interleave){		
		    out_idx_1=cg.n_channels_oversampled*cg.n_rows*proj+cg.n_channels_oversampled*  2*row  +channel;
		    out_idx_2=cg.n_channels_oversampled*cg.n_rows*proj+cg.n_channels_oversampled*(2*row+1)+channel;
		}
		else{
		    out_idx_1=cg.n_channels_oversampled*cg.n_rows*proj+cg.n_channels_oversampled*(2*row+1)+channel;
		    out_idx_2=cg.n_channels_oversampled*cg.n_rows*proj+cg.n_channels_oversampled*  2*row  +channel;
		}
		
		float beta_1 = asin((channel-2*cg.central_channel)*(cg.fan_angle_increment/2)*cg.r_f/r_fr(0.0f,-dr,cg));
		float beta_2 = asin((channel-2*cg.central_channel)*(cg.fan_angle_increment/2)*cg.r_f/r_fr(0.0f,dr,cg));
		float beta_idx_1=get_beta_idx(beta_1,beta_lookup_1,cg.n_channels_oversampled);
		float beta_idx_2=get_beta_idx(beta_2,beta_lookup_2,cg.n_channels_oversampled);
		
		h_output[out_idx_1]=interp3(rebin_t_1,dim,beta_idx_1,row,proj);
		h_output[out_idx_2]=interp3(rebin_t_2,dim,beta_idx_2,row,proj);
		
	    }
	}
    }

    //Copy data into our mr structure, skipping initial truncated projections
    size_t offset=cg.add_projections;
    for (int i=0;i<cg.n_channels_oversampled;i++){
	for (int j=0;j<cg.n_rows;j++){
	    for (int k=0;k<(mr->ri.n_proj_pull/mr->ri.n_ffs-2*cg.add_projections);k++){
		mr->ctd.rebin[k*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i]=h_output[(k+offset)*cg.n_channels_oversampled*cg.n_rows+j*cg.n_channels_oversampled+i];
	    }
	}
    }

    printf("Filtering...\n");
    
    // Load and run filter
    float * h_filter=(float*)calloc(2*cg.n_channels_oversampled,sizeof(float));
    load_filter(h_filter,mr);

    for (int i=0;i<(n_proj-2*cg.add_projections);i++){
	for (int j=0;j<cg.n_rows;j++){
	    int row_start_idx=i*cg.n_channels_oversampled*cg.n_rows+cg.n_channels_oversampled*j;
	    filter_cpu(&mr->ctd.rebin[row_start_idx],h_filter,cg.n_channels_oversampled);
	}
    }
    
    // Check "testing" flag, write rebin to disk if set
    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/rebin.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->ctd.rebin,sizeof(float),cg.n_channels_oversampled*cg.n_rows*(mr->ri.n_proj_pull-2*cg.add_projections_ffs)/mr->ri.n_ffs,outfile);
	fclose(outfile);
    }
    
    free(beta_lookup_1);
    free(beta_lookup_2);
    free(rebin_t_1);
    free(rebin_t_2);    
    free(h_output);
}

void filter_cpu(float * row, float * filter, int N){
    // N is the number of elements in a row

    // Calculate padding
    int M=2*pow(2.0f,ceil(log2((float)N)));

    // Create two new padded/manipulated vectors from our inputs into fftw complex arrays
    fftw_complex * R = (fftw_complex*)fftw_malloc(sizeof(fftw_complex)*M);
    memset(R,0,M*sizeof(fftw_complex));
    for (int i=0;i<N;i++){
	R[i][0]=row[i];
        R[i][1]=0.0;
    }

    fftw_complex * F=(fftw_complex*)fftw_malloc(sizeof(fftw_complex)*M);
    memset(F,0,M*sizeof(fftw_complex));
    
    for (int i=0;i<N;i++){
	F[i][0]=filter[(int)floor((2.0f*N-1.0f)/2.0f)+1+i];
        F[i][1]=0.0;
    }
    for (int i=(M-N+1);i<M;i++){
	F[i][0]=filter[i-(M-N+1)+1];
        F[i][1]=filter[i-(M-N+1)+1];
    }

    // Allocate complex output vectors for row and filter FFTs
    fftw_complex * R_fourier=(fftw_complex*)fftw_malloc(sizeof(fftw_complex)*M);
    fftw_complex * F_fourier=(fftw_complex*)fftw_malloc(sizeof(fftw_complex)*M);

    // Create plans and execute FFTs
    fftw_plan p_R,p_F;
    p_R=fftw_plan_dft_1d(M,R,R_fourier,FFTW_FORWARD,FFTW_ESTIMATE);
    p_F=fftw_plan_dft_1d(M,F,F_fourier,FFTW_FORWARD,FFTW_ESTIMATE);
    fftw_execute(p_R);
    fftw_execute(p_F);

    //Multiply row and filter into output array
    fftw_complex * O_fourier=(fftw_complex*)fftw_malloc(sizeof(fftw_complex)*M);
    for (int i=0;i<M;i++){
      //O_fourier[i]=R_fourier[i]*F_fourier[i];

        O_fourier[i][0] = R_fourier[i][0]*F_fourier[i][0] - R_fourier[i][1]*F_fourier[i][1];
        O_fourier[i][1] = R_fourier[i][0]*F_fourier[i][1] - R_fourier[i][1]*F_fourier[i][0];
        
    }

    //Prep final output array and plan, then execute
    fftw_complex * O=(fftw_complex*)fftw_malloc(sizeof(fftw_complex)*M);
    fftw_plan p_O;
    p_O=fftw_plan_dft_1d(M,O_fourier,O,FFTW_BACKWARD,FFTW_ESTIMATE);
    fftw_execute(p_O);

    //Copy real portion of final result into source row
    for (int i=0;i<N;i++){
      //row[i]=(1.0f/(float)M)*(float)creal(O[i]);
      row[i]=(1.0f/(float)M)*(float)(O[i][0]);
    }

    // Clean up
    fftw_destroy_plan(p_R);
    fftw_destroy_plan(p_F);
    fftw_destroy_plan(p_O);    

    fftw_free(F);
    fftw_free(F_fourier);
    fftw_free(R);
    fftw_free(R_fourier);
    fftw_free(O);
    fftw_free(O_fourier);    
}
