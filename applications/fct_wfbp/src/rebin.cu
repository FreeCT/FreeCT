#include "hip/hip_runtime.h"
#include <rebin.h>
#include <iostream>
#include <util.h>

#include <gpu_error_check.h>
#include <hipfft/hipfft.h>

#include <chrono>

// @JOHN: Don't sweat seconds: focus on clean code and algorithmic improvements

// GPU CODE block
__constant__ struct CTGeometry d_cg;
__constant__ struct ReconConfig d_rp;
texture<float,hipTextureType2D,hipReadModeElementType> tex_row_sheet;

__global__ void rebin_kernel(float * output){

  int channel_idx = threadIdx.x + blockDim.x*blockIdx.x;
  int proj_idx    = threadIdx.y + blockDim.y*blockIdx.y;

  float delta_theta = 2.0f*3.14159265f/(float)d_cg.projections_per_rotation;
  
  float p = ((float)channel_idx - d_cg.detector_central_col)*(float)d_cg.detector_pixel_size_col;
  float beta = asin(p/d_cg.distance_source_to_detector);
  float theta = (float)proj_idx * delta_theta;
  float alpha = theta - beta;
  
  float beta_idx  = (beta*d_cg.distance_source_to_detector/d_cg.detector_pixel_size_col) + d_cg.detector_central_col;
  float alpha_idx = alpha/delta_theta;
  
  int out_idx = channel_idx + proj_idx*d_cg.num_detector_cols;
  output[out_idx] = tex2D(tex_row_sheet, beta_idx + 0.5f, alpha_idx + 0.5);
  //output[out_idx] = d_cg.detector_pixel_size_col; //tex2D(tex_row_sheet, channel_idx + 0.5f, proj_idx + 0.5);

}

__global__ void multiply_filter(hipfftComplex * row_sheet_fourier_domain, float * filter){

  int channel_idx = threadIdx.x + blockDim.x*blockIdx.x;
  int proj_idx    = threadIdx.y + blockDim.y*blockIdx.y;

  int idx = channel_idx + proj_idx * d_cg.num_detector_cols;
  row_sheet_fourier_domain[idx] = filter[channel_idx] * row_sheet_fourier_domain[channel_idx];

}

inline void configure_texture(){
  tex_row_sheet.addressMode[0] = hipAddressModeClamp;
  tex_row_sheet.addressMode[1] = hipAddressModeClamp;
  tex_row_sheet.addressMode[2] = hipAddressModeClamp;
  tex_row_sheet.filterMode     = hipFilterModeLinear;
  tex_row_sheet.normalized     = false;
}

// HOST CODE
void generate_filter(float * f_array,CTGeometry cg, float c = 1.0f, float a = 1.0f);
  
void rebin(std::shared_ptr<float> output, std::shared_ptr<float> input, CTGeometry cg, ReconConfig rp){

  // Reshape the raw data to be "row sheets"
  // Additionally, flip the channel direction and row direction
  // since Chen et al 2015 defines the geometry to be the opposite
  // of how Stierstorfer et al 2004 defines it.
  Timer t;
  t.tic();
  std::cout << "Reshaping raw data array..." << std::endl;
  std::shared_ptr<float> raw_reshaped_ptr(new float[cg.num_detector_cols*cg.num_detector_rows*cg.total_number_of_projections]);
  float * raw = input.get();
  float * raw_reshaped = raw_reshaped_ptr.get();
  for (int i=0; i<cg.total_number_of_projections;i++){
    for (int j=0; j<cg.num_detector_rows;j++){
      for (int k=0; k<cg.num_detector_cols;k++){
        int input_idx = k + j*cg.num_detector_cols + i*cg.num_detector_cols*cg.num_detector_rows;
        int output_idx = (cg.num_detector_cols - 1 - k) + i*cg.num_detector_cols + (cg.num_detector_rows - 1 - j)*cg.num_detector_cols*cg.total_number_of_projections;
        raw_reshaped[output_idx] = raw[input_idx];
      }
    }
  }
  std::cout << "Done!" << std::endl;
  t.toc();

  // Allocate our GPU arrays
  hipError_t gpu_status;
  
  hipChannelFormatDesc channelDesc=hipCreateChannelDesc<float>();
  hipArray * d_row_sheet_raw;
  gpu_status = hipMallocArray(&d_row_sheet_raw, &channelDesc, cg.num_detector_cols, cg.total_number_of_projections);
  gpuErrChk(gpu_status);
  
  float * d_row_sheet_rebin;
  gpu_status = hipMalloc(&d_row_sheet_rebin,cg.num_detector_cols*cg.total_number_of_projections*sizeof(float));
  gpuErrChk(gpu_status);
  
  gpu_status = hipMemcpyToSymbol(HIP_SYMBOL(d_cg),&cg,sizeof(struct CTGeometry),0,hipMemcpyHostToDevice);
  gpuErrChk(gpu_status);
  
  gpu_status = hipMemcpyToSymbol(HIP_SYMBOL(d_rp),&rp,sizeof(struct ReconConfig),0,hipMemcpyHostToDevice);
  gpuErrChk(gpu_status);

  configure_texture();

  // Create our filter and send to GPU
  float * h_filter = new float[]
  
  
  // Main rebin/filter loop
  float * rebinned_data = output.get();
  
  GPUTimer gt;
  for (int i=0; i<cg.num_detector_rows; i++){
    gt.tic();    
    size_t offset = i*cg.num_detector_cols*cg.total_number_of_projections;
    size_t sheet_size_bytes =  cg.num_detector_cols*cg.total_number_of_projections * sizeof(float);
    gpu_status = hipMemcpyToArray(d_row_sheet_raw, 0, 0, &raw_reshaped[offset], sheet_size_bytes, hipMemcpyHostToDevice);
    gpuErrChk(gpu_status);

    gpu_status = hipBindTextureToArray(tex_row_sheet,d_row_sheet_raw,channelDesc);
    gpuErrChk(gpu_status);

    dim3 rebin_threads(8,8);
    dim3 rebin_blocks(cg.num_detector_cols/rebin_threads.x,cg.total_number_of_projections/rebin_threads.y);
    rebin_kernel<<<rebin_blocks,rebin_threads>>>(d_row_sheet_rebin);
    gpuErrChk(hipPeekAtLastError());

    // Configure FFT plan and run filtering
    hipfftHandle plan;
    hipfftComplex * d_sheet_data_fourier_domain;

    int RANK = 1;
    int NX = cg.num_detector_cols;
    int BATCH = cg.total_number_of_projections;
    
    hipMalloc(&d_sheet_data_fourier_domain,cg.num_detector_cols*cg.total_number_of_projections*sizeof(hipfftComplex));

    hipfftPlanMany(&plan,RANK,&NX,
                  NULL,1,0,
                  NULL,1,0,
                  HIPFFT_R2C,BATCH);
    hipfftExecR2C(plan,d_row_sheet_rebin,d_sheet_data_fourier_domain);

    dim3 filter_threads(cg.num_detector_cols,1);
    dim3 filter_blocks(1,cg.total_number_of_projections/rebin_threads.y);
    multiply_filter<<<filter_blocks,filter_threads>>>(d_sheet_data_fourier_domain,);
    
    // Copy data back from GPU
    hipMemcpy(&rebinned_data[offset],d_row_sheet_rebin,sheet_size_bytes,hipMemcpyDeviceToHost);


      
    gt.toc();
    
  }
  
  hipFreeArray(d_row_sheet_raw);
  hipFree(d_row_sheet_rebin);
}

void generate_filter(float * f_array, CTGeometry cg, float c, float a){
  // Create a spatial domain ramp filter.  Eventually we'll expost c and a
  // so users can customize filter response for smoother/sharper reconstructions
  
  //float * h_filter=(float*)calloc(2*cg.n_channels_oversampled,sizeof(float));
  //float ds = mr->cg.r_f*sin(mr->cg.fan_angle_increment/2.0f); // This is at isocenter.  Is that correct?

  float pi_f = 3.141592653589f;
  float ds = cg.detector_pixel_size_col;
  
  //float ds = mr->cg.src_to_det*sin(mr->cg.fan_angle_increment/2.0f); // I think it should be at the detector

  auto r = [](float t)-> float{
             float v = sin(t)/t + (cos(t)-1.0f)/(t*t);
             if (t==0)
               v=0.5;
             return v;
           };
  
  int test = (int)cg.num_detector_cols;
  
  for (int i = -test;i < test;i++){
    
    f_array[i+cg.num_detector_cols] = (c*c/(2.0f*ds)) * (a*r(c*pi_f*i) +
                                                         (((1.0f-a)/2.0f)*r(pi_f*c*i + pi_f)) +
                                                         (((1.0f -a)/2.0f)*r(pi_f*c*i-pi_f)));
    
  }
  
}
