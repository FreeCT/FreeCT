#include "hip/hip_runtime.h"
#include <rebin.h>
#include <iostream>
#include <util.h>

#include <hipfft/hipfft.h>
#include <gpu_error_check.h>

#include <fstream>
#include <chrono>
#include <vector>

// @JOHN: Don't sweat seconds: focus on clean code and algorithmic improvements

// GPU CODE block
__constant__ struct CTGeometry d_cg;
__constant__ struct ReconConfig d_rp;
texture<float,hipTextureType2D,hipReadModeElementType> tex_row_sheet;

__global__ void rebin_kernel(float * output){

  int channel_idx = threadIdx.x + blockDim.x*blockIdx.x;
  int proj_idx    = threadIdx.y + blockDim.y*blockIdx.y;

  float delta_theta = 2.0f*3.14159265f/(float)d_cg.projections_per_rotation;
  
  float p = ((float)channel_idx - d_cg.detector_central_col)*(float)d_cg.detector_pixel_size_col;
  float beta = asin(p/d_cg.distance_source_to_detector);
  float theta = (float)proj_idx * delta_theta;
  float alpha = theta - beta;
  
  float beta_idx  = (beta*d_cg.distance_source_to_detector/d_cg.detector_pixel_size_col) + d_cg.detector_central_col;
  float alpha_idx = alpha/delta_theta;
  
  int out_idx = channel_idx + proj_idx*d_cg.num_detector_cols;
  output[out_idx] = tex2D(tex_row_sheet, beta_idx + 0.5f, alpha_idx + 0.5);
  //output[out_idx] = d_cg.detector_pixel_size_col; //tex2D(tex_row_sheet, channel_idx + 0.5f, proj_idx + 0.5);
}

__global__ void multiply_filter(hipfftComplex * row_sheet_fourier_domain, hipfftComplex * filter){
  int channel_idx = threadIdx.x + blockDim.x*blockIdx.x;
  int proj_idx    = threadIdx.y + blockDim.y*blockIdx.y;

  int idx = channel_idx + proj_idx * d_cg.num_detector_cols;
  //row_sheet_fourier_domain[idx] = filter[channel_idx] * row_sheet_fourier_domain[channel_idx];

  float a = filter[channel_idx].x;
  float b = filter[channel_idx].y;
  float c = row_sheet_fourier_domain[channel_idx].x;
  float d = row_sheet_fourier_domain[channel_idx].y;
  
  row_sheet_fourier_domain[idx].x = a;//a*c - b*d;
  row_sheet_fourier_domain[idx].y = b;//a*d + b*c;
}

inline void configure_texture(){
  tex_row_sheet.addressMode[0] = hipAddressModeClamp;
  tex_row_sheet.addressMode[1] = hipAddressModeClamp;
  tex_row_sheet.addressMode[2] = hipAddressModeClamp;
  tex_row_sheet.filterMode     = hipFilterModeLinear;
  tex_row_sheet.normalized     = false;
}

// HOST CODE
hipfftComplex * generate_filter(CTGeometry cg, float c = 1.0f, float a = 1.0f);
  
void rebin(std::shared_ptr<float> output, std::shared_ptr<float> input, CTGeometry cg, ReconConfig rp){

  // Reshape the raw data to be "row sheets"
  // Additionally, flip the channel direction and row direction
  // since Chen et al 2015 defines the geometry to be the opposite
  // of how Stierstorfer et al 2004 defines it.
  // (May be worth it to eventually do this on the GPU...)
  Timer t;
  t.tic();
  std::cout << "Reshaping raw data array..." << std::endl;
  std::shared_ptr<float> raw_reshaped_ptr(new float[cg.num_detector_cols*cg.num_detector_rows*cg.total_number_of_projections]);
  float * raw = input.get();
  float * raw_reshaped = raw_reshaped_ptr.get();
  for (int i=0; i<cg.total_number_of_projections;i++){
    for (int j=0; j<cg.num_detector_rows;j++){
      for (int k=0; k<cg.num_detector_cols;k++){
        int input_idx = k + j*cg.num_detector_cols + i*cg.num_detector_cols*cg.num_detector_rows;
        int output_idx = (cg.num_detector_cols - 1 - k) + i*cg.num_detector_cols + (cg.num_detector_rows - 1 - j)*cg.num_detector_cols*cg.total_number_of_projections;
        raw_reshaped[output_idx] = raw[input_idx];
      }
    }
  }
  std::cout << "Done!" << std::endl;
  t.toc();

  // Allocate our GPU arrays
  hipError_t gpu_status;
  
  hipChannelFormatDesc channelDesc=hipCreateChannelDesc<float>();
  hipArray * d_row_sheet_raw;
  gpu_status = hipMallocArray(&d_row_sheet_raw, &channelDesc, cg.num_detector_cols, cg.total_number_of_projections);
  gpuErrChk(gpu_status);
  
  float * d_row_sheet_rebin;
  gpu_status = hipMalloc(&d_row_sheet_rebin,cg.num_detector_cols*cg.total_number_of_projections*sizeof(float));
  gpuErrChk(gpu_status);
  
  gpu_status = hipMemcpyToSymbol(HIP_SYMBOL(d_cg),&cg,sizeof(struct CTGeometry),0,hipMemcpyHostToDevice);
  gpuErrChk(gpu_status);
  
  gpu_status = hipMemcpyToSymbol(HIP_SYMBOL(d_rp),&rp,sizeof(struct ReconConfig),0,hipMemcpyHostToDevice);
  gpuErrChk(gpu_status);

  configure_texture();

  // Create our filter (on device, ready to be multiplied against our data)  
  hipfftComplex * d_filter = generate_filter(cg);

  // Create our FFT plans
  hipfftResult cufft_status;
  hipfftHandle plan_forward;
  hipfftHandle plan_reverse;
  
  int RANK  = 1;
  int NX    = cg.num_detector_cols;
  int BATCH = cg.total_number_of_projections;

  cufft_status = hipfftPlanMany(&plan_forward,RANK,&NX,
                               NULL,1,0,
                               NULL,1,0,
                               HIPFFT_R2C,BATCH);
  cufftErrChk(cufft_status);

  cufft_status = hipfftPlanMany(&plan_reverse,RANK,&NX,
                               NULL,1,0,
                               NULL,1,0,
                               HIPFFT_C2R,BATCH);
  cufftErrChk(cufft_status);

  // Allocate FFT result data
  hipfftComplex * d_sheet_data_fourier_domain;
  gpu_status = hipMalloc(&d_sheet_data_fourier_domain,cg.num_detector_cols*cg.total_number_of_projections*sizeof(hipfftComplex));
  gpuErrChk(gpu_status);
    
  // Main rebin/filter loop
  float * rebinned_data = output.get();
  
  GPUTimer gt;
  for (int i=0; i<cg.num_detector_rows; i++){
    gt.tic();

    // Copy raw projection data to texture memory
    size_t offset = i*cg.num_detector_cols*cg.total_number_of_projections;
    size_t sheet_size_bytes =  cg.num_detector_cols*cg.total_number_of_projections * sizeof(float);
    gpu_status = hipMemcpyToArray(d_row_sheet_raw, 0, 0, &raw_reshaped[offset], sheet_size_bytes, hipMemcpyHostToDevice);
    gpuErrChk(gpu_status);

    gpu_status = hipBindTextureToArray(tex_row_sheet,d_row_sheet_raw,channelDesc);
    gpuErrChk(gpu_status);

    // Run rebinning kernel
    dim3 rebin_threads(8,8);
    dim3 rebin_blocks(cg.num_detector_cols/rebin_threads.x,cg.total_number_of_projections/rebin_threads.y);
    rebin_kernel<<<rebin_blocks,rebin_threads>>>(d_row_sheet_rebin);
    gpuErrChk(hipPeekAtLastError());

    // Filter the rebinned data
    cufft_status = hipfftExecR2C(plan_forward,(hipfftReal*)d_row_sheet_rebin,d_sheet_data_fourier_domain);
    cufftErrChk(cufft_status);
    hipDeviceSynchronize();

    dim3 filter_threads(cg.num_detector_cols,1);
    dim3 filter_blocks(1,cg.total_number_of_projections/filter_threads.y);
    multiply_filter<<<filter_blocks,filter_threads>>>(d_sheet_data_fourier_domain,d_filter);
    gpuErrChk(hipPeekAtLastError());
    hipDeviceSynchronize();

    cufft_status = hipfftExecC2R(plan_reverse,d_sheet_data_fourier_domain,(hipfftReal*)d_row_sheet_rebin);
    cufftErrChk(cufft_status);
    hipDeviceSynchronize();
    
    // Copy data back from GPU
    gpu_status = hipMemcpy(&rebinned_data[offset],d_row_sheet_rebin,sheet_size_bytes,hipMemcpyDeviceToHost);
    gpuErrChk(gpu_status);
      
    gt.toc();
  }

  hipfftDestroy(plan_forward);
  hipfftDestroy(plan_reverse);
  
  hipFree(d_sheet_data_fourier_domain);
  hipFree(d_filter);
  hipFreeArray(d_row_sheet_raw);
  hipFree(d_row_sheet_rebin);
}

hipfftComplex * generate_filter(CTGeometry cg, float c, float a){
  // Create a spatial domain ramp filter.  Eventually we'll expost c and a
  // so users can customize filter response for smoother/sharper reconstructions  
  //float ds = mr->cg.r_f*sin(mr->cg.fan_angle_increment/2.0f); // This is at isocenter.  Is that correct?

  std::cout << "Generating filter" << std::endl;

  // Allocate the host filter
  std::shared_ptr<float> h_filter(new float[cg.num_detector_cols]);

  // Calculate the filter
  float pi_f = 3.141592653589f;
  float ds = cg.detector_pixel_size_col;
  
  auto r = [](float t)->float{
             float v = sin(t)/t + (cos(t)-1.0f)/(t*t);
             if (t==0)
               v=0.5;
             return v;
           };

  int N = cg.num_detector_cols;
  
  for (int i=-N/2;i<N/2;i++){    
    h_filter.get()[i+N/2] = (c*c/(2.0f*ds)) * (a*r(c*pi_f*i) +
                                                         (((1.0f-a)/2.0f)*r(pi_f*c*i + pi_f)) +
                                                         (((1.0f -a)/2.0f)*r(pi_f*c*i-pi_f)));
  }

  // Apply the "fftshift" operation
  for (int i=0; i<N/2;i++){
    float tmp = h_filter.get()[i];   
    h_filter.get()[i]     = h_filter.get()[i+N/2];
    h_filter.get()[i+N/2] = tmp;
  }
  
  // Send to device
  hipError_t cuda_status;
  hipfftResult cufft_status;
  
  float * d_filter;
  cuda_status = hipMalloc(&d_filter,cg.num_detector_cols*sizeof(float));
  gpuErrChk(cuda_status);

  cuda_status = hipMemcpy(d_filter,h_filter.get(),cg.num_detector_cols*sizeof(float),hipMemcpyHostToDevice);
  gpuErrChk(cuda_status);

  // Take the FFT to get it into the Fourier domain, and return pointer to the complex FFT array
  hipfftHandle plan;
  cufft_status = hipfftPlan1d(&plan,cg.num_detector_cols,HIPFFT_R2C,1);
  cufftErrChk(cufft_status);
    
  hipfftComplex * d_filter_final;
  cuda_status = hipMalloc(&d_filter_final,cg.num_detector_cols*sizeof(hipfftComplex));
  gpuErrChk(cuda_status);
  
  cufft_status = hipfftExecR2C(plan,(hipfftReal*)d_filter,d_filter_final);
  cufftErrChk(cufft_status);

  hipFree(d_filter);
  hipfftDestroy(plan);

  return d_filter_final;
  
}


