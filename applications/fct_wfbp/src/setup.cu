#include "hip/hip_runtime.h"
/* FreeCT_wFBP is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <sys/types.h>
#include <pwd.h>

#include <setup.h>
#include <parse_config.h>
#include <fct/FreeCTRead.h>

#include <memory>

#define pi 3.1415926535897f
#define BLOCK_SLICES 32

void split_path_file(char**p, char**f, char *pf);
int array_search(float key,double * array,int numel_array,int search_type);
void remove_trailing_slash(char * str);

int configure_paths(struct recon_metadata *mr){
    
    /* --- Get working directory and User's home directory --- */
    struct passwd *pw=getpwuid(getuid());    
    //const char * home_dir=pw->pw_dir;
    //strcpy(mr->home_dir,home_dir);
    getcwd(mr->cwd,4096*sizeof(char));

    /* --- Get where the executable is running ---*/
    char full_exe_path[4096]={0};
    char * exe_path=(char*)calloc(4096,sizeof(char));
    char * exe_name=(char*)calloc(255,sizeof(char));
    readlink("/proc/self/exe",full_exe_path,4096);
    split_path_file(&exe_path,&exe_name,full_exe_path);
    strcpy(mr->install_dir,exe_path);
    mr->install_dir[strlen(mr->install_dir)-1]=0;

    /* --- Check for defined output path ---*/
    // if not defined, set to current working directory
    if (strcmp(mr->rp.output_dir,"")==0)
	strcpy(mr->rp.output_dir,mr->cwd);

    /* --- Check for output file name --- */
    // if not defined, set to rawdatafile.img
    if(strcmp(mr->rp.output_file,"")==0){
	char fullpath[4096+255]={0};
	sprintf(fullpath,"%s.img",mr->rp.raw_data_file);
	strcpy(mr->rp.output_file,fullpath);
    }

    // Cleanup directory strings
    //remove_trailing_slash(mr->home_dir);
    remove_trailing_slash(mr->install_dir);
    remove_trailing_slash(mr->cwd);    
    remove_trailing_slash(mr->rp.output_dir);
    remove_trailing_slash(mr->rp.raw_data_dir);    

    /* Check to make sure we can read the raw data file */
    char fullpath[4096+255]={0};
    FILE * fid;    
    memset(fullpath,0,4096+255);
    sprintf(fullpath,"%s/%s",mr->rp.raw_data_dir,mr->rp.raw_data_file);
    
    fid=fopen(fullpath,"r");
    if (fid==NULL){
	return 1;
    }
    else{
    	fclose(fid);
    }
    
    /* Check to make sure we can write to output file */
    memset(fullpath,0,4096+255);
    sprintf(fullpath,"%s/%s",mr->rp.output_dir,mr->rp.output_file);

    fid=fopen(fullpath,"w");
    if (fid==NULL){
	return 2;
    }
    else{
    	fclose(fid);
	// Theres a better way to do this... but for now this works
	remove(fullpath);	
    }
    
    return 0;
}

struct recon_params configure_recon_params(char * filename){
    struct recon_params prms;
    memset(&prms, 0,sizeof(prms));
    
    parse_config(filename,&prms);

    // Convert our table_dir_str to our table_dir integer
    if (strcmp(prms.table_dir_str,"")!=0){
	if (strcmp(prms.table_dir_str,"out")==0){
	    prms.table_dir=1;
	}
	else if (strcmp(prms.table_dir_str,"in")==0){
	    prms.table_dir=-1;
	}
	else{
	    printf("WARNING: TableDir parameter must be 'in' or 'out' (no quotes).  Defaulting to 'out'.\n");
	    prms.table_dir=1;
	}
    }

    // Perform some sanity checks to make sure that we have read in the "essentials"
    // Bail if critical values are zero
    int exit_flag=0;
    //if (prms.n_rows==0){
    //    printf("Nrows was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    //if (prms.coll_slicewidth==0){
    //    printf("CollSlicewidth was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    if (prms.slice_thickness==0){
	printf("SliceThickness was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    //if (prms.pitch_value==0){
    //    printf("PitchValue was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    //if (prms.acq_fov==0){
    //    printf("AcqFOV was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    if (prms.recon_fov==0){
	printf("ReconFOV was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    //if (prms.n_readings==0){
    //    printf("Readings was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    if (prms.nx==0){
	printf("Nx was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    if (prms.ny==0){
	printf("Ny was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    //if (prms.file_type==0&&prms.table_dir==0){
    //    printf("WARNING: 'TableDir' parameter unset.  Defaulting to 'out'.\n");
    //    prms.table_dir=1;
    //}
    if (exit_flag){
	exit(1);
    }
    
    return prms; 
} 

struct ct_geom configure_ct_geom(struct recon_metadata *mr){ 

    struct ct_geom cg;
    memset(&cg,0,sizeof(cg));

    // Runtime polymorphism to eventually support multiple raw data formats
    std::string raw_data_path = mr->rp.raw_data_dir;
    std::unique_ptr<fct::RawDataSet> ds = std::make_unique<fct::DicomDataSet>();
    ds->setPath(raw_data_path);
    ds->initialize();
    ds->readMetadata();

    // Physical geometry of the scanner (cannot change from scan to scan)
    // TCIA/Mayo clinic format does not automatically account for FFS in projections per rotation
    // Number of detector rows
    cg.anode_angle=7.0f*pi/180.0f; //!!!!!! How to get this for GE scanners?  Maybe we don't need it?
    cg.r_f             = ds->getDistSourceToIsocenter(); 
    cg.src_to_det      = ds->getDistSourceToDetector();
    cg.central_channel = ds->getDetectorCentralChannel();
    cg.n_rows          = ds->getDetectorRows();
    cg.n_channels      = ds->getDetectorChannels();

    if (ds->getFlyingFocalSpotMode()=="FFSNONE"){
      mr->rp.phi_ffs = 0;
      mr->rp.z_ffs   = 0;
    }
    else if (ds->getFlyingFocalSpotMode()=="FFSXY"){
      mr->rp.phi_ffs = 1;
      mr->rp.z_ffs   = 0;
    }

    else if (ds->getFlyingFocalSpotMode()=="FFSZ"){
      mr->rp.phi_ffs = 0;
      mr->rp.z_ffs   = 1;
    }
    else if (ds->getFlyingFocalSpotMode()=="FFSXYZ"){
      mr->rp.phi_ffs = 1;
      mr->rp.z_ffs   = 1;
    }
    else {
      std::cout << "ERROR: Unsupported flying focal spot mode!" << std::endl;
      exit(1);
    }

    cg.n_proj_turn = ds->getProjectionsPerRotation();
    cg.n_proj_ffs  = cg.n_proj_turn*pow(2,mr->rp.phi_ffs)*pow(2,mr->rp.z_ffs);
    cg.n_rows_raw  = cg.n_rows;     //(unsigned int)(rp.n_rows/pow(2,rp.z_ffs));
    cg.n_rows      = cg.n_rows*pow(2,mr->rp.z_ffs);
    
    cg.fan_angle_increment = atan(ds->getDetectorTransverseSpacing()/ds->getDistSourceToDetector());
    mr->rp.coll_slicewidth = ds->getDistSourceToIsocenter()*(ds->getDetectorAxialSpacing()/ds->getDistSourceToDetector());

    //cg.z_rot = 
    ////cg.z_rot               = rp.pitch_value;  
    //cg.add_projections     = (cg.fan_angle_increment*cg.n_channels/2)/(2.0f*pi/cg.n_proj_turn)+10; 
    //cg.add_projections_ffs = cg.add_projections*pow(2,rp.z_ffs)*pow(2,rp.phi_ffs);
    //
    //
    //cg.theta_cone=2.0f*atan(7.5f*1.2f/cg.r_f); 
    
    //char * cg_buffer;
    //char * token;
    //
    //cg.table_direction=rp.table_dir;
    //
    //char path[4096+255];
    //int scanner=-1;
    //// First attempt to parse scanner as full filepath    
    //FILE * cg_file;
    //cg_file=fopen(mr->rp.scanner,"r");
    //if (cg_file==NULL){
    //    // Next attempt to find the file in the "resources" directory of the project
    //    strcpy(path,mr->install_dir);
    //    strcat(path,"/resources/scanners/");
    //    strcat(path,mr->rp.scanner);
    //    cg_file=fopen(path,"r");
    //    if (cg_file==NULL){
    //        // Finally, try it as a number for a hardcoded scanner 
    //        scanner=atoi(mr->rp.scanner);
    //        if ((scanner<0||scanner>2)||(scanner==0&&(strlen(mr->rp.scanner)!=1))){
    //    	perror("Could not parse selected scanner");
    //    	exit(1);
    //        }
    //        
    //        // If we use a hardcoded scanner, we want to supercede file subtype
    //        switch (scanner){
    //        case 0:// binary files, don't care
    //    	break;
    //        case 1:// DefinitionAS -> filetype=ptr -> 1 
    //    	mr->rp.file_subtype=1;
    //    	break;
    //        case 2:// Sensation64 -> filetype=ctd -> 2
    //    	mr->rp.file_subtype=2;
    //    	break;
    //        }
    //    }
    //}
    //
    //if (scanner==-1){// Found scanner file
    //    fseek(cg_file, 0, SEEK_END);
    //    size_t cg_size = ftell(cg_file);
    //    rewind(cg_file);
    //    cg_buffer = (char*)malloc(cg_size + 1);
    //    cg_buffer[cg_size] = '\0';
    //    fread(cg_buffer, sizeof(char), cg_size, cg_file);
    //    fclose(cg_file);
    //
    //    token=strtok(cg_buffer," \t\n%");
    //
    //    //Parse parameter file
    //    while (token!=NULL){
    //        if (strcmp(token,"RSrcToIso:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%f",&cg.r_f);
    //        }
    //        else if (strcmp(token,"RSrcToDet:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%f",&cg.src_to_det);
    //        }
    //        else if (strcmp(token,"AnodeAngle:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%f",&cg.anode_angle);
    //        }
    //        else if (strcmp(token,"FanAngleInc:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%f",&cg.fan_angle_increment);
    //        }
    //        else if (strcmp(token,"ThetaCone:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%f",&cg.theta_cone);
    //        }
    //        else if (strcmp(token,"CentralChannel:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%f",&cg.central_channel);
    //        }
    //        else if (strcmp(token,"NProjTurn:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%lu",&cg.n_proj_turn);
    //        }
    //        else if (strcmp(token,"NChannels:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%lu",&cg.n_channels);
    //        }
    //        else if (strcmp(token,"ReverseRowInterleave:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%i",&cg.reverse_row_interleave);
    //        }
    //        else if (strcmp(token,"ReverseChanInterleave:")==0){
    //    	token=strtok(NULL," \t\n%");
    //    	sscanf(token,"%i",&cg.reverse_channel_interleave);
    //        }
    //        else { 
    //    	//token=strtok(NULL," \t\n%"); 
    //        }
    //        
    //        token=strtok(NULL," \t\n%"); 
    //    }
    //    
    //    free(cg_buffer);
    //}
    //
    //// If we did not parse from a file, and have a valid number for
    //// the scanner, get our ct_geom from the hardcoded options
    //switch (scanner){ 
    //case -1:
    //    // Finish everything out
    //    cg.n_proj_ffs=cg.n_proj_turn*pow(2,rp.phi_ffs)*pow(2,rp.z_ffs); 
    //    cg.n_channels_oversampled=2*cg.n_channels;
    //    cg.n_rows=(unsigned int)rp.n_rows;
    //    cg.n_rows_raw=(unsigned int)(rp.n_rows/pow(2,rp.z_ffs));
    //    cg.z_rot=rp.pitch_value;
    //    cg.add_projections=(cg.fan_angle_increment*cg.n_channels/2)/(2.0f*pi/cg.n_proj_turn)+10; 	
    //    cg.add_projections_ffs=cg.add_projections*pow(2,rp.z_ffs)*pow(2,rp.phi_ffs);
    //    
    //    break;
    //case 0: // Non-standard scanner (in this case Fred Noo's Simulated Scanner)
    //
    //        //float det_spacing_1=1.4083f;
    //        //float det_spacing_2=1.3684f;
    //     
    //        // Physical geometry of the scanner (cannot change from scan to scan) 
    //    cg.r_f=570.0f; 
    //    cg.src_to_det=1040.0f; 
    //    cg.anode_angle=7.0f*pi/180.0f; 
    //    cg.fan_angle_increment=1.4083f/cg.src_to_det;
    //    cg.theta_cone=2.0f*atan(7.5f*1.3684f/cg.src_to_det);
    //    cg.central_channel=335.25f; 
    //
    //    // Size and setup of the detector helix 
    //    cg.n_proj_turn=1160; 
    //    cg.n_proj_ffs=cg.n_proj_turn*pow(2,rp.phi_ffs)*pow(2,rp.z_ffs); 
    //    cg.n_channels=672; 
    //    cg.n_channels_oversampled=2*cg.n_channels; 
    //    cg.n_rows=(unsigned int)rp.n_rows; 
    //    cg.n_rows_raw=(unsigned int)(rp.n_rows/pow(2,rp.z_ffs)); 
    //    cg.z_rot=rp.pitch_value;
    //    cg.add_projections=(cg.fan_angle_increment*cg.n_channels/2)/(2.0f*pi/cg.n_proj_turn)+10; 
    //    cg.add_projections_ffs=cg.add_projections*pow(2,rp.z_ffs)*pow(2,rp.phi_ffs); 
    //
    //    break; 
    //
    //case 1: // Definition AS 
    //    
    //        // Physical geometry of the scanner (cannot change from scan to scan) 
    //    cg.r_f=595.0f; 
    //    cg.src_to_det=1085.6f; 
    //    cg.anode_angle=7.0f*pi/180.0f; 
    //    cg.fan_angle_increment=0.067864f*pi/180.0f; 
    //    cg.theta_cone=2.0f*atan(7.5f*1.2f/cg.r_f); 
    //    cg.central_channel=366.25f;
    //
    //    // Size and setup of the detector helix 
    //    cg.n_proj_turn=1152; 
    //    cg.n_proj_ffs=cg.n_proj_turn*pow(2,rp.phi_ffs)*pow(2,rp.z_ffs); 
    //    cg.n_channels=736; 
    //    cg.n_channels_oversampled=2*cg.n_channels; 
    //    cg.n_rows=(unsigned int)rp.n_rows; 
    //    cg.n_rows_raw=(unsigned int)(rp.n_rows/pow(2,rp.z_ffs)); 
    //    cg.z_rot=rp.pitch_value;
    //    cg.add_projections=(cg.fan_angle_increment*cg.n_channels/2)/(2.0f*pi/cg.n_proj_turn)+10; 
    //    cg.add_projections_ffs=cg.add_projections*pow(2,rp.z_ffs)*pow(2,rp.phi_ffs); 
    //    
    //    break; 
    //
    //case 2: // Sensation 64 
    //
    //        // Physical geometry of the scanner (cannot change from scan to scan) 
    //    cg.r_f=570.0f; 
    //    cg.src_to_det=1040.0f; 
    //    //cg.anode_angle=12.0f*pi/180.0f;
    //    cg.anode_angle=7.0f*pi/180.0f;
    //    cg.fan_angle_increment=0.07758621f*pi/180.0f;
    //    //cg.theta_cone=2.0f*atan(7.5f*1.2f/cg.r_f);
    //    cg.theta_cone=2.0f*atan(7.5f*1.2f/cg.r_f); 	
    //    cg.central_channel=334.25f; 
    //
    //    // Size and setup of the detector helix 
    //    cg.n_proj_turn=1160; 
    //    cg.n_proj_ffs=cg.n_proj_turn*pow(2,rp.phi_ffs)*pow(2,rp.z_ffs); 
    //    cg.n_channels=672; 
    //    cg.n_channels_oversampled=2*cg.n_channels; 
    //    cg.n_rows=(unsigned int)rp.n_rows; 
    //    cg.n_rows_raw=(unsigned int)(rp.n_rows/pow(2,rp.z_ffs)); 
    //    cg.z_rot=rp.pitch_value;
    //    cg.add_projections=(cg.fan_angle_increment*cg.n_channels/2)/(2.0f*pi/cg.n_proj_turn)+10; 
    //    cg.add_projections_ffs=cg.add_projections*pow(2,rp.z_ffs)*pow(2,rp.phi_ffs); 
    //
    //    break; 
    //} 
    //
    //cg.acq_fov=rp.acq_fov; 
    //
    //if (rp.phi_ffs==1){
    //    cg.central_channel=floor(cg.central_channel)+0.375f;
    //    //cg.central_channel+=0.375f; 
    //}
    
    return cg;
}

void configure_reconstruction(struct recon_metadata *mr){
    /* --- Get tube angles and table positions --- */
    struct ct_geom cg=mr->cg;
    struct recon_params rp=mr->rp;

    // Allocate the memory
    mr->tube_angles=(float*)calloc(rp.n_readings,sizeof(float));
    mr->table_positions=(double*)calloc(rp.n_readings,sizeof(double));
    
    char fullpath[4096+255]={0};
    sprintf(fullpath,"%s/%s",rp.raw_data_dir,rp.raw_data_file);
    
    FILE * raw_file;
    raw_file=fopen(fullpath,"rb");
    if (raw_file==NULL){
	perror("Raw data file not found.");
	exit(1);	
    }
    
    //switch (rp.file_type){
    //case 0:{; // Binary file
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=fmod(((360.0f/cg.n_proj_ffs)*i+rp.tube_start_angle),360.0f);
    //    	if (cg.table_direction==-1)
    //    	    mr->table_positions[i]=((float)rp.n_readings/(float)cg.n_proj_ffs)*cg.z_rot-(float)i*cg.z_rot/(float)cg.n_proj_ffs;
    //    	else if (cg.table_direction==1)
    //    	    mr->table_positions[i]=0.0f+(float)i*cg.z_rot/(float)cg.n_proj_ffs;
    //    	else 
    //    	    mr->table_positions[i]=0.0f+(float)i*cg.z_rot/(float)cg.n_proj_ffs;
    //        }	
    //        break;}
    //case 1:{; //DefinitionAS Raw
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadPTRTubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=((double)ReadPTRTablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw))/1000.0;		
    //        }
    //        
    //        // Clean up the table positions because they tend to
    //        // be wonky at the ends when read directly from the
    //        // raw data
    //    	
    //        // <0 is decreasing table position >0 is increasing
    //        int direction=(mr->table_positions[100]-mr->table_positions[0])/fabs(mr->table_positions[100]-mr->table_positions[0]);
    //        
    //        for (int i=1;i<rp.n_readings;i++){
    //    	mr->table_positions[i]=mr->table_positions[0]+(double)cg.z_rot*(((double)i)/(pow(2.0,rp.z_ffs)*pow(2.0,rp.phi_ffs)*(double)cg.n_proj_turn))*(double)direction;
    //        }
    //
    //        break;}
    //case 2:{; //CTD v1794 (Pre 2015 Sensation64)
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadCTDv1794TubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadCTDv1794TablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //case 3:{; //CTD v2007 (Post 2015 Sensation64)
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadCTDv2007TubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadCTDv2007TablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //case 4:{; //IMA (can wrap any of the above (except binary)
    //        int raw_data_subtype=mr->rp.file_subtype; // Determine if we're looking for PTR or CTD
    //    
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadIMATubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw,raw_data_subtype,rp.raw_data_offset);
    //    	mr->table_positions[i]=((double)ReadIMATablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw,raw_data_subtype,rp.raw_data_offset))/1000.0;
    //        }
    //
    //        // Clean up the table positions because they tend to
    //        // be wonky at the ends when read directly from the
    //        // raw data
    //
    //        // <0 is decreasing table position >0 is increasing
    //        int direction=(mr->table_positions[100]-mr->table_positions[0])/fabs(mr->table_positions[100]-mr->table_positions[0]);
    //        
    //        for (int i=1;i<rp.n_readings;i++){
    //    	mr->table_positions[i]=mr->table_positions[0]+(double)cg.z_rot*(((double)i)/(pow(2.0,rp.z_ffs)*pow(2.0,rp.phi_ffs)*(double)cg.n_proj_turn))*(double)direction;
    //        }
    //        
    //        break;}
    //case 5:{; //Force Raw
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadForceTubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadForceTablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //case 6:{; //DICOM Raw
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadDICOMTubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadDICOMTablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //}
    fclose(raw_file);

    /* --- Figure out how many and which projections to grab --- */

    int n_ffs=pow(2,rp.z_ffs)*pow(2,rp.phi_ffs);
    int n_slices_block=BLOCK_SLICES;

    int recon_direction=fabs(rp.end_pos-rp.start_pos)/(rp.end_pos-rp.start_pos);
    if (recon_direction!=1&&recon_direction!=-1) // user request one slice (end_pos==start_pos)
	recon_direction=1;

    // override end_pos if user has set the number of slices
    if (rp.n_slices!=0){
	rp.end_pos=rp.start_pos+(rp.n_slices-1)*rp.slice_thickness;
    }
    
    float recon_start_pos = rp.start_pos - recon_direction*rp.slice_thickness;
    float recon_end_pos   = rp.end_pos   + recon_direction*rp.slice_thickness;//rp.start_pos+recon_direction*(n_slices_recon-1)*rp.coll_slicewidth;

    int n_slices_requested=floor(fabs(recon_end_pos-recon_start_pos)/rp.coll_slicewidth)+1;//floor(fabs(rp.end_pos-rp.start_pos)/rp.coll_slicewidth)+1;
    int n_slices_recon=(n_slices_requested-1)+(n_slices_block-(n_slices_requested-1)%n_slices_block);

    recon_end_pos=recon_start_pos+recon_direction*(n_slices_recon-1)*rp.coll_slicewidth;
    
    int n_blocks=n_slices_recon/n_slices_block;

    //float recon_start_pos=rp.start_pos;
    //float recon_end_pos=rp.start_pos+recon_direction*(n_slices_recon-1)*rp.coll_slicewidth;
    int array_direction=fabs(mr->table_positions[100]-mr->table_positions[0])/(mr->table_positions[100]-mr->table_positions[0]);
    int idx_slice_start=array_search(recon_start_pos,mr->table_positions,rp.n_readings,array_direction);
    int idx_slice_end=array_search(recon_end_pos,mr->table_positions,rp.n_readings,array_direction);

    // Decide if the user has requested a valid range for reconstruction
    mr->ri.data_begin_pos = mr->table_positions[0];
    mr->ri.data_end_pos   = mr->table_positions[rp.n_readings-1];
    float projection_padding= cg.z_rot * (cg.n_proj_ffs/2+cg.add_projections_ffs+256)/cg.n_proj_ffs;
    float allowed_begin = mr->ri.data_begin_pos+array_direction*projection_padding;
    float allowed_end   = mr->ri.data_end_pos-array_direction*projection_padding;

    mr->ri.allowed_begin = allowed_begin;
    mr->ri.allowed_end   = allowed_end;

    // Check "testing" flag, write raw to disk if set
    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/table_positions.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->table_positions,sizeof(double),rp.n_readings,outfile);
	fclose(outfile);

	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/tube_angles.ct_test");
	outfile=fopen(fullpath,"w");
	fwrite(mr->tube_angles,sizeof(float),rp.n_readings,outfile);
	fclose(outfile);
    }

    if (((rp.start_pos>allowed_begin)&&(rp.start_pos>allowed_end))||((rp.start_pos<allowed_begin)&&(rp.start_pos<allowed_end))){
	printf("Requested reconstruction is outside of allowed data range: %.2f to %.2f\n",allowed_begin,allowed_end);
	exit(1);
    }
    
    if (((rp.end_pos>allowed_begin)&&(rp.end_pos>allowed_end))||((rp.end_pos<allowed_begin)&&(rp.end_pos<allowed_end))){
	printf("Requested reconstruction is outside of allowed data range: %.2f to %.2f\n",allowed_begin,allowed_end);
	exit(1);
    }

    // We always pull projections in the order they occur in the raw
    // data.  If the end_pos comes before the start position in the
    // array, we use the end_pos as the "first" slice to pull
    // projections for.  This method will take into account the
    // ordering of projections with ascending or descending table
    // position, as well as any slice ordering the user requests.
    
    int idx_pull_start;
    int idx_pull_end;

    int pre_post_buffer=cg.n_proj_ffs/2;
    if (rp.z_ffs==1){
	pre_post_buffer=cg.n_proj_ffs/2;
    }
    
    if (idx_slice_start>idx_slice_end){
	idx_pull_start=idx_slice_end-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_slice_start+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }
    else{
	idx_pull_start=idx_slice_start-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_slice_end+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }

    idx_pull_end+=256;
   
    int n_proj_pull=idx_pull_end-idx_pull_start;
    
    // Ensure that we have a number of projections divisible by 128 (because GPU)
    n_proj_pull=(n_proj_pull-1)+(128-(n_proj_pull-1)%128);
    idx_pull_end=idx_pull_start+n_proj_pull;
    
    // copy this info into our recon metadata
    mr->cg.table_direction=array_direction;
    mr->rp.end_pos=rp.end_pos;
    mr->ri.n_ffs=n_ffs;
    mr->ri.n_slices_requested=n_slices_requested;
    mr->ri.n_slices_recon=n_slices_recon;
    mr->ri.n_slices_block=n_slices_block;
    mr->ri.n_blocks=n_blocks;
    mr->ri.idx_slice_start=idx_slice_start;
    mr->ri.idx_slice_end=idx_slice_end; 
    mr->ri.recon_start_pos=recon_start_pos;
    mr->ri.recon_end_pos=recon_end_pos;;
    mr->ri.idx_pull_start=idx_pull_start;
    mr->ri.idx_pull_end=idx_pull_end;
    mr->ri.n_proj_pull=n_proj_pull;

    /* --- Allocate our raw data array and our rebin array --- */
    mr->ctd.raw=(float*)calloc(cg.n_channels*cg.n_rows_raw*n_proj_pull,sizeof(float));
    mr->ctd.rebin=(float*)calloc(cg.n_channels_oversampled*cg.n_rows*(n_proj_pull-2*cg.add_projections_ffs)/n_ffs,sizeof(float));
    mr->ctd.image=(float*)calloc(rp.nx*rp.ny*n_slices_recon,sizeof(float));
}

void update_block_info(recon_metadata *mr){

    struct recon_info ri=mr->ri;
    struct recon_params rp=mr->rp;
    struct ct_geom cg=mr->cg;

    free(mr->ctd.raw);
    free(mr->ctd.rebin);
    
    /* --- Figure out how many and which projections to grab --- */
    int n_ffs=pow(2,rp.z_ffs)*pow(2,rp.phi_ffs);

    int recon_direction=fabs(rp.end_pos-rp.start_pos)/(rp.end_pos-rp.start_pos);
    if (recon_direction!=1&&recon_direction!=-1) // user requests one slice (end_pos==start_pos)
	recon_direction=1;
    
    float block_slice_start=ri.recon_start_pos+recon_direction*ri.cb.block_idx*rp.coll_slicewidth*(float)ri.n_slices_block;
    float block_slice_end=block_slice_start+(float)recon_direction*((float)ri.n_slices_block-1.0f)*rp.coll_slicewidth;
    int array_direction=fabs(mr->table_positions[100]-mr->table_positions[0])/(mr->table_positions[100]-mr->table_positions[0]);
    int idx_block_slice_start=array_search(block_slice_start,mr->table_positions,rp.n_readings,array_direction);
    int idx_block_slice_end=array_search(block_slice_end,mr->table_positions,rp.n_readings,array_direction);

    // We always pull projections in the order they occur in the raw
    // data.  If the end_pos comes before the start position in the
    // array, we use the end_pos as the "first" slice to pull
    // projections for.  This method will take into account the
    // ordering of projections with ascending or descending table
    // position, as well as any slice ordering the user requests.
    
    int idx_pull_start;
    int idx_pull_end;

    int pre_post_buffer=cg.n_proj_ffs/2;
    if (rp.z_ffs==1){
	pre_post_buffer=cg.n_proj_ffs/2;
    }

    if (idx_block_slice_start>idx_block_slice_end){
	idx_pull_start=idx_block_slice_end-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_block_slice_start+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }
    else{
	idx_pull_start=idx_block_slice_start-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_block_slice_end+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }

    idx_pull_end+=256;
   
    int n_proj_pull=idx_pull_end-idx_pull_start;

    // Ensure that we have a number of projections divisible by 128 (because GPU)
    n_proj_pull=(n_proj_pull-1)+(128-(n_proj_pull-1)%128);
    idx_pull_end=idx_pull_start+n_proj_pull;
    
    // copy this info into our recon metadata
    mr->ri.cb.block_slice_start=block_slice_start;
    mr->ri.cb.block_slice_end=block_slice_end;
    mr->ri.cb.idx_block_slice_start=idx_block_slice_start;
    mr->ri.cb.idx_block_slice_end=idx_block_slice_end; 

    mr->ri.idx_pull_start=idx_pull_start;
    mr->ri.idx_pull_end=idx_pull_end;
    mr->ri.n_proj_pull=n_proj_pull;

    mr->ri.cb.block_idx++;

    // Reallocate our raw and rebin arrays to account for changing n_proj_pull
    mr->ctd.raw=(float*)calloc(cg.n_channels*cg.n_rows_raw*n_proj_pull,sizeof(float));
    mr->ctd.rebin=(float*)calloc(cg.n_channels_oversampled*cg.n_rows*(n_proj_pull-2*cg.add_projections_ffs)/n_ffs,sizeof(float));
    
}

void extract_projections(struct recon_metadata * mr){

    float * frame_holder=(float*)calloc(mr->cg.n_channels*mr->cg.n_rows_raw,sizeof(float));

    FILE * raw_file;
    struct recon_params rp=mr->rp;
    struct ct_geom cg=mr->cg;
    char fullpath[4096+255]={0};
    sprintf(fullpath,"%s/%s",rp.raw_data_dir,rp.raw_data_file);
    raw_file=fopen(fullpath,"rb");
    
    //switch (mr->rp.file_type){
    //case 0:{ // binary
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadBinaryFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder,mr->rp.raw_data_offset);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 1:{ // DefinitionAS
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadPTRFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 2:{ // CTD v1794 
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadCTDv1794Frame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 3:{ // CTD v2007
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadCTDv2007Frame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 4:{ // IMA (wraps either PTR or IMA)
    //    int raw_data_subtype=rp.file_subtype;
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadIMAFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder,raw_data_subtype,rp.raw_data_offset);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}	
    //case 5:{ //Force Raw
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        
    //        ReadForceFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //
    //    }
    //    break;}
    //case 6:{ //DICOM Raw
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadDICOMFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //}

    // Check "testing" flag, write raw to disk if set
    if (mr->flags.testing){
	memset(fullpath,0,4096+255);
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/raw.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->ctd.raw,sizeof(float),cg.n_channels*cg.n_rows_raw*mr->ri.n_proj_pull,outfile);
	fclose(outfile);
    }
    
    fclose(raw_file);
    free(frame_holder);
}

void finish_and_cleanup(struct recon_metadata * mr){

    int n_slices_final=floor(fabs(mr->rp.end_pos-mr->rp.start_pos)/mr->rp.slice_thickness)+1;
    
    // Write the image data to disk
    char fullpath[4096+255]={0};
    sprintf(fullpath,"%s/%s",mr->rp.output_dir,mr->rp.output_file);
    FILE * outfile=fopen(fullpath,"w");
    fwrite(mr->ctd.final_image_stack,sizeof(float),mr->rp.nx*mr->rp.ny*n_slices_final,outfile);
    fclose(outfile);

    // Free all remaining allocations in metadata
    free(mr->ctd.rebin);
    free(mr->ctd.image);
    free(mr->ctd.raw);
    free(mr->ctd.final_image_stack);    
    free(mr->tube_angles);
    free(mr->table_positions);
}


void remove_trailing_slash(char * str){
    size_t len=strlen(str);
    if ((len>0)&&(str[len-1]=='/')){
	str[len-1]='\0';
    }
}

void split_path_file(char**p, char**f, char *pf) {
    char *slash = pf, *next;
    while ((next = strpbrk(slash + 1, "\\/"))) slash = next;
    if (pf != slash) slash++;
    *p = strndup(pf, slash - pf);
    *f = strdup(slash);
}


int array_search(float key,double * array,int numel_array,int search_type){
    int idx=0;

    switch (search_type){
    case -1:{// Array descending
	while (key<array[idx]&&idx<numel_array){
	    idx++;}
	break;}
    case 0:{// Find where we're equal
	while (key!=array[idx]&&idx<numel_array){
	    idx++;}
	break;}
    case 1:{// Array ascending
	while (key>array[idx]&&idx<numel_array){
	    idx++;}
	break;}
    }

    return idx;
}
