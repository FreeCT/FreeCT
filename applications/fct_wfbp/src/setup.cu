#include "hip/hip_runtime.h"
/* FreeCT_wFBP is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <sys/types.h>
#include <pwd.h>

#include <setup.h>
#include <parse_config.h>
#include <fct/FreeCTRead.h>

#include <memory>

#define pi 3.1415926535897f
#define BLOCK_SLICES 32

void split_path_file(char**p, char**f, char *pf);
int array_search(float key,double * array,int numel_array,int search_type);
void remove_trailing_slash(char * str);

int configure_paths(struct recon_metadata *mr){
    
    /* --- Get working directory and User's home directory --- */
    struct passwd *pw=getpwuid(getuid());    
    //const char * home_dir=pw->pw_dir;
    //strcpy(mr->home_dir,home_dir);
    getcwd(mr->cwd,4096*sizeof(char));

    /* --- Get where the executable is running ---*/
    char full_exe_path[4096]={0};
    char * exe_path=(char*)calloc(4096,sizeof(char));
    char * exe_name=(char*)calloc(255,sizeof(char));
    readlink("/proc/self/exe",full_exe_path,4096);
    split_path_file(&exe_path,&exe_name,full_exe_path);
    strcpy(mr->install_dir,exe_path);
    mr->install_dir[strlen(mr->install_dir)-1]=0;

    /* --- Check for defined output path ---*/
    // if not defined, set to current working directory
    if (strcmp(mr->rp.output_dir,"")==0)
	strcpy(mr->rp.output_dir,mr->cwd);

    /* --- Check for output file name --- */
    // if not defined, set to rawdatafile.img
    if(strcmp(mr->rp.output_file,"")==0){
	char fullpath[4096+255]={0};
	sprintf(fullpath,"%s.img",mr->rp.raw_data_file);
	strcpy(mr->rp.output_file,fullpath);
    }

    // Cleanup directory strings
    //remove_trailing_slash(mr->home_dir);
    remove_trailing_slash(mr->install_dir);
    remove_trailing_slash(mr->cwd);    
    remove_trailing_slash(mr->rp.output_dir);
    remove_trailing_slash(mr->rp.raw_data_dir);    

    /* Check to make sure we can read the raw data file */
    char fullpath[4096+255]={0};
    FILE * fid;    
    memset(fullpath,0,4096+255);
    sprintf(fullpath,"%s/%s",mr->rp.raw_data_dir,mr->rp.raw_data_file);
    
    fid=fopen(fullpath,"r");
    if (fid==NULL){
	return 1;
    }
    else{
    	fclose(fid);
    }
    
    /* Check to make sure we can write to output file */
    memset(fullpath,0,4096+255);
    sprintf(fullpath,"%s/%s",mr->rp.output_dir,mr->rp.output_file);

    fid=fopen(fullpath,"w");
    if (fid==NULL){
	return 2;
    }
    else{
    	fclose(fid);
	// Theres a better way to do this... but for now this works
	remove(fullpath);	
    }
    
    return 0;
}

struct recon_params configure_recon_params(char * filename){
    struct recon_params prms;
    memset(&prms, 0,sizeof(prms));
    
    parse_config(filename,&prms);

    // Convert our table_dir_str to our table_dir integer
    if (strcmp(prms.table_dir_str,"")!=0){
	if (strcmp(prms.table_dir_str,"out")==0){
	    prms.table_dir=1;
	}
	else if (strcmp(prms.table_dir_str,"in")==0){
	    prms.table_dir=-1;
	}
	else{
	    printf("WARNING: TableDir parameter must be 'in' or 'out' (no quotes).  Defaulting to 'out'.\n");
	    prms.table_dir=1;
	}
    }

    // Perform some sanity checks to make sure that we have read in the "essentials"
    // Bail if critical values are zero
    int exit_flag=0;
    //if (prms.n_rows==0){
    //    printf("Nrows was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    //if (prms.coll_slicewidth==0){
    //    printf("CollSlicewidth was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    if (prms.slice_thickness==0){
	printf("SliceThickness was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    //if (prms.pitch_value==0){
    //    printf("PitchValue was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    //if (prms.acq_fov==0){
    //    printf("AcqFOV was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    if (prms.recon_fov==0){
	printf("ReconFOV was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    //if (prms.n_readings==0){
    //    printf("Readings was not properly set in configuration.  Check parameter file.\n");
    //    exit_flag=1;
    //}
    if (prms.nx==0){
	printf("Nx was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    if (prms.ny==0){
	printf("Ny was not properly set in configuration.  Check parameter file.\n");
	exit_flag=1;
    }
    //if (prms.file_type==0&&prms.table_dir==0){
    //    printf("WARNING: 'TableDir' parameter unset.  Defaulting to 'out'.\n");
    //    prms.table_dir=1;
    //}
    if (exit_flag){
	exit(1);
    }
    
    return prms; 
} 

struct ct_geom configure_ct_geom(struct recon_metadata *mr){ 

    struct ct_geom cg;
    memset(&cg,0,sizeof(cg));

    // Runtime polymorphism to eventually support multiple raw data formats
    std::string raw_data_path = mr->rp.raw_data_dir;
    std::unique_ptr<fct::RawDataSet> ds = std::make_unique<fct::DicomDataSet>();
    ds->setPath(raw_data_path);
    ds->initialize();
    ds->readMetadata();

    // Physical geometry of the scanner (cannot change from scan to scan)
    // TCIA/Mayo clinic format does not automatically account for FFS in projections per rotation
    // Number of detector rows
    cg.anode_angle=7.0f*pi/180.0f; //!!!!!! How to get this for GE scanners?  Maybe we don't need it?
    cg.r_f             = ds->getDistSourceToIsocenter(); 
    cg.src_to_det      = ds->getDistSourceToDetector();
    cg.central_channel = ds->getDetectorCentralChannel();
    cg.n_rows          = ds->getDetectorRows();
    cg.n_channels      = ds->getDetectorChannels();
    cg.n_channels_oversampled = 2 * ds->getDetectorChannels();
    
    if (ds->getFlyingFocalSpotMode()=="FFSNONE"){
      mr->rp.phi_ffs = 0;
      mr->rp.z_ffs   = 0;
    }
    else if (ds->getFlyingFocalSpotMode()=="FFSXY"){
      mr->rp.phi_ffs = 1;
      mr->rp.z_ffs   = 0;
    }

    else if (ds->getFlyingFocalSpotMode()=="FFSZ"){
      mr->rp.phi_ffs = 0;
      mr->rp.z_ffs   = 1;
    }
    else if (ds->getFlyingFocalSpotMode()=="FFSXYZ"){
      mr->rp.phi_ffs = 1;
      mr->rp.z_ffs   = 1;
    }
    else {
      std::cout << "ERROR: Unsupported flying focal spot mode!" << std::endl;
      exit(1);
    }

    cg.n_proj_turn = ds->getProjectionsPerRotation();
    cg.n_proj_ffs = ds->getProjectionsPerRotation();
    std::cout << "Unclear if number of projections in Mayo Clinic format accounts for FFS!" << std::endl;
    //cg.n_proj_ffs  = cg.n_proj_turn*pow(2,mr->rp.phi_ffs)*pow(2,mr->rp.z_ffs);
    cg.n_rows_raw  = cg.n_rows;     //(unsigned int)(rp.n_rows/pow(2,rp.z_ffs));
    cg.n_rows      = cg.n_rows*pow(2,mr->rp.z_ffs);
    
    cg.fan_angle_increment = atan(ds->getDetectorTransverseSpacing()/ds->getDistSourceToDetector());
    mr->rp.coll_slicewidth = ds->getDistSourceToIsocenter()*(ds->getDetectorAxialSpacing()/ds->getDistSourceToDetector());

    // To accurately compute z_rot (table feed per rotation in mm)
    // we load 0->(n_rot) and read compute the travel
    ds->readProjection(0);
    ds->readProjection(cg.n_proj_ffs);

    cg.z_rot = ds->getTablePosition(cg.n_proj_ffs) - ds->getTablePosition(0);

    cg.add_projections     = (cg.fan_angle_increment*cg.n_channels/2)/(2.0f*pi/cg.n_proj_turn)+10; 
    cg.add_projections_ffs = cg.add_projections*pow(2,mr->rp.z_ffs)*pow(2,mr->rp.phi_ffs);

    //cg.theta_cone=2.0f*atan(7.5f*1.2f/cg.r_f);
    float detector_cone_offset = ((float)(cg.n_rows - 1))/2.0f;
    cg.theta_cone=2.0f*atan(detector_cone_offset * mr->rp.coll_slicewidth/cg.r_f);

    cg.acq_fov = 2.0f * cg.r_f*sin((float(cg.n_channels-1)/2.0) * ds->getDetectorTransverseSpacing() * (1.0f/cg.src_to_det));

    cg.table_direction = cg.z_rot/fabs(cg.z_rot);
    cg.z_rot = fabs(cg.z_rot);

    std::cout << "CT Geometry and Scan derived parameters: " << std::endl;
    std::cout << "===========================================" << std::endl;
    std::cout << "Num projections per turn:          "       << cg.n_proj_turn            << std::endl;
    std::cout << "Num projections per turn (ffs):    "       << cg.n_proj_ffs             << std::endl;
    std::cout << "Num detector channels:             "       << cg.n_channels             << std::endl;
    std::cout << "Num detector channels (ffs):       "       << cg.n_channels_oversampled << std::endl;
    std::cout << "Num detector rows:                 "       << cg.n_rows_raw             << std::endl;
    std::cout << "Num detector rows (ffs):           "       << cg.n_rows                 << std::endl;
    std::cout << "Radius src->isocenter (mm):        "       << cg.r_f                    << std::endl;
    std::cout << "Radius src->detector (mm):         "       << cg.src_to_det             << std::endl;
    std::cout << "Table feed per rotation (mm):      "       << cg.z_rot                  << std::endl;
    std::cout << "Theta cone (rad):                  "       << cg.theta_cone             << std::endl;
    std::cout << "Fan angle increment (rad):         "       << cg.fan_angle_increment    << std::endl;
    std::cout << "Anode angle (rad) *:               "       << cg.anode_angle            << std::endl;
    std::cout << "Central channel:                   "       << cg.central_channel        << std::endl;
    std::cout << "Acquisition FOV (mm):              "       << cg.acq_fov                << std::endl;
    std::cout << "Projection block buffer:           "       << cg.add_projections        << std::endl;
    std::cout << "Projection block buffer (ffs):     "       << cg.add_projections_ffs    << std::endl;
    std::cout << "Table direction *:                 "       << cg.table_direction        << std::endl;
    std::cout << "Collimated slicewidth @ isocenter: "       << mr->rp.coll_slicewidth    << std::endl;
    
    return cg;
}

void configure_reconstruction(struct recon_metadata *mr){
    /* --- Get tube angles and table positions --- */
    struct ct_geom cg=mr->cg;
    struct recon_params rp=mr->rp;

    // Allocate the memory
    mr->tube_angles=(float*)calloc(rp.n_readings,sizeof(float));
    mr->table_positions=(double*)calloc(rp.n_readings,sizeof(double));
    
    char fullpath[4096+255]={0};
    sprintf(fullpath,"%s/%s",rp.raw_data_dir,rp.raw_data_file);
    
    FILE * raw_file;
    raw_file=fopen(fullpath,"rb");
    if (raw_file==NULL){
	perror("Raw data file not found.");
	exit(1);	
    }
    
    //switch (rp.file_type){
    //case 0:{; // Binary file
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=fmod(((360.0f/cg.n_proj_ffs)*i+rp.tube_start_angle),360.0f);
    //    	if (cg.table_direction==-1)
    //    	    mr->table_positions[i]=((float)rp.n_readings/(float)cg.n_proj_ffs)*cg.z_rot-(float)i*cg.z_rot/(float)cg.n_proj_ffs;
    //    	else if (cg.table_direction==1)
    //    	    mr->table_positions[i]=0.0f+(float)i*cg.z_rot/(float)cg.n_proj_ffs;
    //    	else 
    //    	    mr->table_positions[i]=0.0f+(float)i*cg.z_rot/(float)cg.n_proj_ffs;
    //        }	
    //        break;}
    //case 1:{; //DefinitionAS Raw
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadPTRTubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=((double)ReadPTRTablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw))/1000.0;		
    //        }
    //        
    //        // Clean up the table positions because they tend to
    //        // be wonky at the ends when read directly from the
    //        // raw data
    //    	
    //        // <0 is decreasing table position >0 is increasing
    //        int direction=(mr->table_positions[100]-mr->table_positions[0])/fabs(mr->table_positions[100]-mr->table_positions[0]);
    //        
    //        for (int i=1;i<rp.n_readings;i++){
    //    	mr->table_positions[i]=mr->table_positions[0]+(double)cg.z_rot*(((double)i)/(pow(2.0,rp.z_ffs)*pow(2.0,rp.phi_ffs)*(double)cg.n_proj_turn))*(double)direction;
    //        }
    //
    //        break;}
    //case 2:{; //CTD v1794 (Pre 2015 Sensation64)
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadCTDv1794TubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadCTDv1794TablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //case 3:{; //CTD v2007 (Post 2015 Sensation64)
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadCTDv2007TubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadCTDv2007TablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //case 4:{; //IMA (can wrap any of the above (except binary)
    //        int raw_data_subtype=mr->rp.file_subtype; // Determine if we're looking for PTR or CTD
    //    
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadIMATubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw,raw_data_subtype,rp.raw_data_offset);
    //    	mr->table_positions[i]=((double)ReadIMATablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw,raw_data_subtype,rp.raw_data_offset))/1000.0;
    //        }
    //
    //        // Clean up the table positions because they tend to
    //        // be wonky at the ends when read directly from the
    //        // raw data
    //
    //        // <0 is decreasing table position >0 is increasing
    //        int direction=(mr->table_positions[100]-mr->table_positions[0])/fabs(mr->table_positions[100]-mr->table_positions[0]);
    //        
    //        for (int i=1;i<rp.n_readings;i++){
    //    	mr->table_positions[i]=mr->table_positions[0]+(double)cg.z_rot*(((double)i)/(pow(2.0,rp.z_ffs)*pow(2.0,rp.phi_ffs)*(double)cg.n_proj_turn))*(double)direction;
    //        }
    //        
    //        break;}
    //case 5:{; //Force Raw
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadForceTubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadForceTablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //case 6:{; //DICOM Raw
    //        for (int i=0;i<rp.n_readings;i++){
    //    	mr->tube_angles[i]=ReadDICOMTubeAngle(raw_file,i,cg.n_channels,cg.n_rows_raw);
    //    	mr->table_positions[i]=(double)ReadDICOMTablePosition(raw_file,i,cg.n_channels,cg.n_rows_raw)/1000.0;
    //        }
    //        break;}
    //}
    fclose(raw_file);

    /* --- Figure out how many and which projections to grab --- */

    int n_ffs=pow(2,rp.z_ffs)*pow(2,rp.phi_ffs);
    int n_slices_block=BLOCK_SLICES;

    int recon_direction=fabs(rp.end_pos-rp.start_pos)/(rp.end_pos-rp.start_pos);
    if (recon_direction!=1&&recon_direction!=-1) // user request one slice (end_pos==start_pos)
	recon_direction=1;

    // override end_pos if user has set the number of slices
    if (rp.n_slices!=0){
	rp.end_pos=rp.start_pos+(rp.n_slices-1)*rp.slice_thickness;
    }
    
    float recon_start_pos = rp.start_pos - recon_direction*rp.slice_thickness;
    float recon_end_pos   = rp.end_pos   + recon_direction*rp.slice_thickness;//rp.start_pos+recon_direction*(n_slices_recon-1)*rp.coll_slicewidth;

    int n_slices_requested=floor(fabs(recon_end_pos-recon_start_pos)/rp.coll_slicewidth)+1;//floor(fabs(rp.end_pos-rp.start_pos)/rp.coll_slicewidth)+1;
    int n_slices_recon=(n_slices_requested-1)+(n_slices_block-(n_slices_requested-1)%n_slices_block);

    recon_end_pos=recon_start_pos+recon_direction*(n_slices_recon-1)*rp.coll_slicewidth;
    
    int n_blocks=n_slices_recon/n_slices_block;

    //float recon_start_pos=rp.start_pos;
    //float recon_end_pos=rp.start_pos+recon_direction*(n_slices_recon-1)*rp.coll_slicewidth;
    int array_direction=fabs(mr->table_positions[100]-mr->table_positions[0])/(mr->table_positions[100]-mr->table_positions[0]);
    int idx_slice_start=array_search(recon_start_pos,mr->table_positions,rp.n_readings,array_direction);
    int idx_slice_end=array_search(recon_end_pos,mr->table_positions,rp.n_readings,array_direction);

    // Decide if the user has requested a valid range for reconstruction
    mr->ri.data_begin_pos = mr->table_positions[0];
    mr->ri.data_end_pos   = mr->table_positions[rp.n_readings-1];
    float projection_padding= cg.z_rot * (cg.n_proj_ffs/2+cg.add_projections_ffs+256)/cg.n_proj_ffs;
    float allowed_begin = mr->ri.data_begin_pos+array_direction*projection_padding;
    float allowed_end   = mr->ri.data_end_pos-array_direction*projection_padding;

    mr->ri.allowed_begin = allowed_begin;
    mr->ri.allowed_end   = allowed_end;

    // Check "testing" flag, write raw to disk if set
    if (mr->flags.testing){
	char fullpath[4096+255];
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/table_positions.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->table_positions,sizeof(double),rp.n_readings,outfile);
	fclose(outfile);

	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/tube_angles.ct_test");
	outfile=fopen(fullpath,"w");
	fwrite(mr->tube_angles,sizeof(float),rp.n_readings,outfile);
	fclose(outfile);
    }

    if (((rp.start_pos>allowed_begin)&&(rp.start_pos>allowed_end))||((rp.start_pos<allowed_begin)&&(rp.start_pos<allowed_end))){
	printf("Requested reconstruction is outside of allowed data range: %.2f to %.2f\n",allowed_begin,allowed_end);
	exit(1);
    }
    
    if (((rp.end_pos>allowed_begin)&&(rp.end_pos>allowed_end))||((rp.end_pos<allowed_begin)&&(rp.end_pos<allowed_end))){
	printf("Requested reconstruction is outside of allowed data range: %.2f to %.2f\n",allowed_begin,allowed_end);
	exit(1);
    }

    // We always pull projections in the order they occur in the raw
    // data.  If the end_pos comes before the start position in the
    // array, we use the end_pos as the "first" slice to pull
    // projections for.  This method will take into account the
    // ordering of projections with ascending or descending table
    // position, as well as any slice ordering the user requests.
    
    int idx_pull_start;
    int idx_pull_end;

    int pre_post_buffer=cg.n_proj_ffs/2;
    if (rp.z_ffs==1){
	pre_post_buffer=cg.n_proj_ffs/2;
    }
    
    if (idx_slice_start>idx_slice_end){
	idx_pull_start=idx_slice_end-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_slice_start+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }
    else{
	idx_pull_start=idx_slice_start-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_slice_end+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }

    idx_pull_end+=256;
   
    int n_proj_pull=idx_pull_end-idx_pull_start;
    
    // Ensure that we have a number of projections divisible by 128 (because GPU)
    n_proj_pull=(n_proj_pull-1)+(128-(n_proj_pull-1)%128);
    idx_pull_end=idx_pull_start+n_proj_pull;
    
    // copy this info into our recon metadata
    mr->cg.table_direction=array_direction;
    mr->rp.end_pos=rp.end_pos;
    mr->ri.n_ffs=n_ffs;
    mr->ri.n_slices_requested=n_slices_requested;
    mr->ri.n_slices_recon=n_slices_recon;
    mr->ri.n_slices_block=n_slices_block;
    mr->ri.n_blocks=n_blocks;
    mr->ri.idx_slice_start=idx_slice_start;
    mr->ri.idx_slice_end=idx_slice_end; 
    mr->ri.recon_start_pos=recon_start_pos;
    mr->ri.recon_end_pos=recon_end_pos;;
    mr->ri.idx_pull_start=idx_pull_start;
    mr->ri.idx_pull_end=idx_pull_end;
    mr->ri.n_proj_pull=n_proj_pull;

    /* --- Allocate our raw data array and our rebin array --- */
    mr->ctd.raw=(float*)calloc(cg.n_channels*cg.n_rows_raw*n_proj_pull,sizeof(float));
    mr->ctd.rebin=(float*)calloc(cg.n_channels_oversampled*cg.n_rows*(n_proj_pull-2*cg.add_projections_ffs)/n_ffs,sizeof(float));
    mr->ctd.image=(float*)calloc(rp.nx*rp.ny*n_slices_recon,sizeof(float));
}

void update_block_info(recon_metadata *mr){

    struct recon_info ri=mr->ri;
    struct recon_params rp=mr->rp;
    struct ct_geom cg=mr->cg;

    free(mr->ctd.raw);
    free(mr->ctd.rebin);
    
    /* --- Figure out how many and which projections to grab --- */
    int n_ffs=pow(2,rp.z_ffs)*pow(2,rp.phi_ffs);

    int recon_direction=fabs(rp.end_pos-rp.start_pos)/(rp.end_pos-rp.start_pos);
    if (recon_direction!=1&&recon_direction!=-1) // user requests one slice (end_pos==start_pos)
	recon_direction=1;
    
    float block_slice_start=ri.recon_start_pos+recon_direction*ri.cb.block_idx*rp.coll_slicewidth*(float)ri.n_slices_block;
    float block_slice_end=block_slice_start+(float)recon_direction*((float)ri.n_slices_block-1.0f)*rp.coll_slicewidth;
    int array_direction=fabs(mr->table_positions[100]-mr->table_positions[0])/(mr->table_positions[100]-mr->table_positions[0]);
    int idx_block_slice_start=array_search(block_slice_start,mr->table_positions,rp.n_readings,array_direction);
    int idx_block_slice_end=array_search(block_slice_end,mr->table_positions,rp.n_readings,array_direction);

    // We always pull projections in the order they occur in the raw
    // data.  If the end_pos comes before the start position in the
    // array, we use the end_pos as the "first" slice to pull
    // projections for.  This method will take into account the
    // ordering of projections with ascending or descending table
    // position, as well as any slice ordering the user requests.
    
    int idx_pull_start;
    int idx_pull_end;

    int pre_post_buffer=cg.n_proj_ffs/2;
    if (rp.z_ffs==1){
	pre_post_buffer=cg.n_proj_ffs/2;
    }

    if (idx_block_slice_start>idx_block_slice_end){
	idx_pull_start=idx_block_slice_end-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_block_slice_start+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }
    else{
	idx_pull_start=idx_block_slice_start-pre_post_buffer-cg.add_projections_ffs;
	idx_pull_start=(idx_pull_start-1)+(n_ffs-(idx_pull_start-1)%n_ffs);
	idx_pull_end=idx_block_slice_end+pre_post_buffer+cg.add_projections_ffs;
	idx_pull_end=(idx_pull_end-1)+(n_ffs-(idx_pull_end-1)%n_ffs);
    }

    idx_pull_end+=256;
   
    int n_proj_pull=idx_pull_end-idx_pull_start;

    // Ensure that we have a number of projections divisible by 128 (because GPU)
    n_proj_pull=(n_proj_pull-1)+(128-(n_proj_pull-1)%128);
    idx_pull_end=idx_pull_start+n_proj_pull;
    
    // copy this info into our recon metadata
    mr->ri.cb.block_slice_start=block_slice_start;
    mr->ri.cb.block_slice_end=block_slice_end;
    mr->ri.cb.idx_block_slice_start=idx_block_slice_start;
    mr->ri.cb.idx_block_slice_end=idx_block_slice_end; 

    mr->ri.idx_pull_start=idx_pull_start;
    mr->ri.idx_pull_end=idx_pull_end;
    mr->ri.n_proj_pull=n_proj_pull;

    mr->ri.cb.block_idx++;

    // Reallocate our raw and rebin arrays to account for changing n_proj_pull
    mr->ctd.raw=(float*)calloc(cg.n_channels*cg.n_rows_raw*n_proj_pull,sizeof(float));
    mr->ctd.rebin=(float*)calloc(cg.n_channels_oversampled*cg.n_rows*(n_proj_pull-2*cg.add_projections_ffs)/n_ffs,sizeof(float));
    
}

void extract_projections(struct recon_metadata * mr){

    float * frame_holder=(float*)calloc(mr->cg.n_channels*mr->cg.n_rows_raw,sizeof(float));

    FILE * raw_file;
    struct recon_params rp=mr->rp;
    struct ct_geom cg=mr->cg;
    char fullpath[4096+255]={0};
    sprintf(fullpath,"%s/%s",rp.raw_data_dir,rp.raw_data_file);
    raw_file=fopen(fullpath,"rb");
    
    //switch (mr->rp.file_type){
    //case 0:{ // binary
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadBinaryFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder,mr->rp.raw_data_offset);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 1:{ // DefinitionAS
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadPTRFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 2:{ // CTD v1794 
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadCTDv1794Frame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 3:{ // CTD v2007
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadCTDv2007Frame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //case 4:{ // IMA (wraps either PTR or IMA)
    //    int raw_data_subtype=rp.file_subtype;
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadIMAFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder,raw_data_subtype,rp.raw_data_offset);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}	
    //case 5:{ //Force Raw
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        
    //        ReadForceFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //
    //    }
    //    break;}
    //case 6:{ //DICOM Raw
    //    for (int i=0;i<mr->ri.n_proj_pull;i++){
    //        ReadDICOMFrame(raw_file,mr->ri.idx_pull_start+i,cg.n_channels,cg.n_rows_raw,frame_holder);
    //        for (int j=0;j<cg.n_channels*cg.n_rows_raw;j++){
    //    	mr->ctd.raw[j+cg.n_channels*cg.n_rows_raw*i]=frame_holder[j];
    //        }
    //    }
    //    break;}
    //}

    // Check "testing" flag, write raw to disk if set
    if (mr->flags.testing){
	memset(fullpath,0,4096+255);
	strcpy(fullpath,mr->rp.output_dir);
	strcat(fullpath,"/raw.ct_test");
	FILE * outfile=fopen(fullpath,"w");
	fwrite(mr->ctd.raw,sizeof(float),cg.n_channels*cg.n_rows_raw*mr->ri.n_proj_pull,outfile);
	fclose(outfile);
    }
    
    fclose(raw_file);
    free(frame_holder);
}

void finish_and_cleanup(struct recon_metadata * mr){

    int n_slices_final=floor(fabs(mr->rp.end_pos-mr->rp.start_pos)/mr->rp.slice_thickness)+1;
    
    // Write the image data to disk
    char fullpath[4096+255]={0};
    sprintf(fullpath,"%s/%s",mr->rp.output_dir,mr->rp.output_file);
    FILE * outfile=fopen(fullpath,"w");
    fwrite(mr->ctd.final_image_stack,sizeof(float),mr->rp.nx*mr->rp.ny*n_slices_final,outfile);
    fclose(outfile);

    // Free all remaining allocations in metadata
    free(mr->ctd.rebin);
    free(mr->ctd.image);
    free(mr->ctd.raw);
    free(mr->ctd.final_image_stack);    
    free(mr->tube_angles);
    free(mr->table_positions);
}


void remove_trailing_slash(char * str){
    size_t len=strlen(str);
    if ((len>0)&&(str[len-1]=='/')){
	str[len-1]='\0';
    }
}

void split_path_file(char**p, char**f, char *pf) {
    char *slash = pf, *next;
    while ((next = strpbrk(slash + 1, "\\/"))) slash = next;
    if (pf != slash) slash++;
    *p = strndup(pf, slash - pf);
    *f = strdup(slash);
}


int array_search(float key,double * array,int numel_array,int search_type){
    int idx=0;

    switch (search_type){
    case -1:{// Array descending
	while (key<array[idx]&&idx<numel_array){
	    idx++;}
	break;}
    case 0:{// Find where we're equal
	while (key!=array[idx]&&idx<numel_array){
	    idx++;}
	break;}
    case 1:{// Array ascending
	while (key>array[idx]&&idx<numel_array){
	    idx++;}
	break;}
    }

    return idx;
}
