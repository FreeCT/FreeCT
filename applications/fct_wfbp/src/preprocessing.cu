#include "hip/hip_runtime.h"
/* CTBangBang is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/

#include <stdlib.h>
#include <stdio.h>
#include <ctbb_macros.h>
#include <recon_structs.h>
#include <preprocessing.cuh>
#include <preprocessing.h>
#include <math.h>


/* Adaptive filtering from Kachelreiss and Kalendar 2001 */
int adaptive_filter_kk(struct recon_metadata * mr){

//    int N=5;
//    for (int i=0; i<N; i++){
//	for (int j=0; j<N; j++){
//	    for (int k=0; k<N; k++){
//		int T_ij=(ceil(N/2)-fabs(i-ceil(N/2)))+(ceil(N/2)-fabs(j-ceil(N/2)))+(ceil(N/2)-fabs(k-ceil(N/2)))+1; 
//		printf("%d,",T_ij);
//	    }
//	    printf("\n");
//	}
//	printf("\n\n");
//    }
//    exit(0);
    
    // Save some typing
    struct ct_geom cg=mr->cg;
    struct recon_info ri=mr->ri;

    // Copy of ct geometry
    hipMemcpyToSymbol(HIP_SYMBOL(d_cg),&cg,sizeof(struct ct_geom),0,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_ri),&ri,sizeof(struct recon_info),0,hipMemcpyHostToDevice);

    // Determine maximum of each projection
    float * d_sup;
    float * h_sup=(float*)calloc(ri.n_proj_pull,sizeof(float));
    hipMalloc(&d_sup,ri.n_proj_pull*sizeof(float));

    float *d_raw;
    hipMalloc(&d_raw,cg.n_channels*cg.n_rows_raw*ri.n_proj_pull*sizeof(float));
    hipMemcpy(d_raw,mr->ctd.raw,cg.n_channels*cg.n_rows_raw*ri.n_proj_pull*sizeof(float),hipMemcpyHostToDevice);

    dim3 extract_sup_threads(128,1,1); // we're guaranteed to get an round number of blocks this way
    dim3 extract_sup_blocks(ri.n_proj_pull/128,1,1);

    extract_sup<<<extract_sup_blocks,extract_sup_threads>>>(d_raw,d_sup);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );


    //hipMemcpy(h_sup,d_sup,ri.n_proj_pull*sizeof(float),hipMemcpyDeviceToHost);
    //float_debug(h_sup,ri.n_proj_pull,"/home/john/Desktop/h_sup.txt");
    
    // Smooth the array of maxima
    float * d_sup_smooth;
    float * h_sup_smooth=(float*)calloc(ri.n_proj_pull,sizeof(float));
    hipMalloc(&d_sup_smooth,ri.n_proj_pull*sizeof(float));
    
    dim3 smooth_sup_threads(128,1,1);
    dim3 smooth_sup_blocks(ri.n_proj_pull/128,1,1);

    smooth_sup<<<extract_sup_blocks,extract_sup_threads>>>(d_sup,d_sup_smooth);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //hipMemcpy(h_sup_smooth,d_sup_smooth,ri.n_proj_pull*sizeof(float),hipMemcpyDeviceToHost);
    //float_debug(h_sup_smooth,ri.n_proj_pull,"/home/john/Desktop/h_sup_smooth.txt");

    // Calculate eccentricity as a function of projection idx
    float * d_ecc;
    float * h_ecc=(float*)calloc(ri.n_proj_pull,sizeof(float));
    hipMalloc(&d_ecc,ri.n_proj_pull*sizeof(float));

    float * d_p_max;
    float * d_p_min;
    hipMalloc(&d_p_max,ri.n_proj_pull*sizeof(float));
    hipMalloc(&d_p_min,ri.n_proj_pull*sizeof(float));
    
    dim3 ecc_threads(128,1,1);
    dim3 ecc_blocks(ri.n_proj_pull/128,1,1);

    eccentricity<<<ecc_blocks,ecc_threads>>>(d_sup_smooth,d_ecc,d_p_max,d_p_min);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );


    //hipMemcpy(h_ecc,d_ecc,ri.n_proj_pull*sizeof(float),hipMemcpyDeviceToHost);
    //float_debug(h_ecc,ri.n_proj_pull,"/home/john/Desktop/h_ecc_trunc.txt");

    // Find thresholds
    float * d_threshold;
    float * h_threshold=(float *)calloc(ri.n_proj_pull,sizeof(float));
    hipMalloc(&d_threshold,ri.n_proj_pull*sizeof(float));

    dim3 threshold_threads(128,1,1);
    dim3 threshold_blocks(ri.n_proj_pull/128,1,1);
    
    find_thresholds<<<threshold_blocks,threshold_threads>>>(mr->rp.adaptive_filtration_s,d_ecc,d_sup_smooth,d_p_max,d_p_min,d_threshold);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    
    //hipMemcpy(h_threshold,d_threshold,ri.n_proj_pull*sizeof(float),hipMemcpyDeviceToHost);
    //float_debug(h_threshold,ri.n_proj_pull,"/home/john/Desktop/h_threshold.txt");

    // Filter the raw projection data
    float * d_filtered_raw;
    hipMalloc(&d_filtered_raw,cg.n_channels*cg.n_rows_raw*ri.n_proj_pull*sizeof(float));
    
    dim3 filter_threads(128,1,1);
    dim3 filter_blocks(ri.n_proj_pull/128,1,1);
    
    filter_projections<<<filter_blocks,filter_threads>>>(d_raw,d_threshold,d_filtered_raw);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Copy filtered raw data back into raw ct data array
    hipMemcpy(mr->ctd.raw,d_filtered_raw,cg.n_channels*cg.n_rows_raw*ri.n_proj_pull*sizeof(float),hipMemcpyDeviceToHost);
    
    //float_debug(mr->ctd.raw,cg.n_channels*cg.n_rows_raw*ri.n_proj_pull,"/home/john/Desktop/h_raw.txt");
    
    hipFree(d_sup);
    hipFree(d_sup_smooth);
    hipFree(d_ecc);
    hipFree(d_p_max);
    hipFree(d_p_min);
    hipFree(d_threshold);
    hipFree(d_raw);
    hipFree(d_filtered_raw);
    free(h_sup);
    free(h_sup_smooth);
    free(h_threshold);
    free(h_ecc);
	       
    return 0;
}
