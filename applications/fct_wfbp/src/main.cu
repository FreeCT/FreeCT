#include "hip/hip_runtime.h"
/* FreeCT_wFBP is GPU and CPU CT reconstruction Software */
/* Copyright (C) 2015  John Hoffman */

/* This program is free software; you can redistribute it and/or */
/* modify it under the terms of the GNU General Public License */
/* as published by the Free Software Foundation; either version 2 */
/* of the License, or (at your option) any later version. */

/* This program is distributed in the hope that it will be useful, */
/* but WITHOUT ANY WARRANTY; without even the implied warranty of */
/* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the */
/* GNU General Public License for more details. */

/* You should have received a copy of the GNU General Public License */
/* along with this program; if not, write to the Free Software */
/* Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA. */

/* Questions and comments should be directed to */
/* jmhoffman@mednet.ucla.edu with "CTBANGBANG" in the subject line*/


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <regex.h>
#include <cstdarg>
#include <unistd.h>
#include <sys/types.h>
#include <pwd.h>

#include <iostream>

#include <ctbb_macros.h>
#include <fct/FreeCTRead.h>
#include <recon_structs.h>
#include <setup.h>
#include <preprocessing.h>
#include <rebin_filter.h>
#include <rebin_filter_cpu.h>
#include <backproject.h>
#include <backproject_cpu.h>
#include <finalize_image_stack.h>
#include <finalize_image_stack_cpu.h>

void log(int verbosity, const char *string, ...);

void usage(){
    printf("\n");
    printf("usage: fct_wfbp [options] input_prm_file\n\n");
    printf("    Options:\n");
    printf("          -v: verbose.\n");
    printf("          -t: test files will be written to desktop.\n");
    printf("    --no-gpu: run program exclusively on CPU. Will override --device=i option.\n");
    printf("  --device=i: run on GPU device number 'i'\n");
    printf("    --timing: Display timing information for each step of the recon process\n");
    printf(" --benchmark: Writes timing data to file used by benchmarking tool\n");
    printf("\n");
    printf("Copyright John Hoffman 2015\n\n");
    exit(0);
}

inline int exists(char * fname){
    int exists;
    if (access(fname,F_OK)!=-1)
        exists=1;
    else
        exists=0;

    return exists;    
}

int main(int argc, char ** argv){

    struct recon_metadata mr;
    memset(&mr,0,sizeof(struct recon_metadata));

    // Parse any command line arguments
    if (argc<2)
	usage();
    
    regex_t regex_dev;
    regmatch_t regmatch_dev;
    if (regcomp(&regex_dev,"--device=*",0)!=0){
	printf("Regex didn't work properly\n");
	exit(1);
    }
    
    for (int i=1;i<(argc-1);i++){

	if (strcmp(argv[i],"-t")==0){
	    mr.flags.testing=1;
	}
	else if (strcmp(argv[i],"-v")==0){
	    mr.flags.verbose=1;
	}
	else if (strcmp(argv[i],"--no-gpu")==0){
	    mr.flags.no_gpu=1;
	}
	else if (regexec(&regex_dev,argv[i],1,&regmatch_dev,0)==0){
	    mr.flags.set_device=1;
	    sscanf(argv[i],"--device=%d",&mr.flags.device_number);
	}
	else if (strcmp(argv[i],"--timing")==0){
	    mr.flags.timing=1;
	}
	else if (strcmp(argv[i],"--benchmark")==0){
	    mr.flags.benchmark=1;
	}
	else if (strcmp(argv[i],"--empty-prm")==0){
	    empty_config(argv[argc-1]);
	}
	else{
	    usage();
	}
    }

    log(mr.flags.verbose,"\n-------------------------\n"
                         "|      FreeCT_wFBP      |\n"
                         "-------------------------\n\n");

    log(mr.flags.verbose,"CHECKING INPUT PARAMETERS AND CONFIGURING RECONSTRUCTION\n"
	"\n");

    /* --- Step 0: configure our processor (CPU or GPU) */
    // We want to send to the GPU furthest back in the list which is
    // unlikely to have a display connected.  We also check for the
    // user passing a specific device number via the command line
    int device_count=0;
    hipGetDeviceCount(&device_count);
    if (device_count==0){
	mr.flags.no_gpu=1;
    }
    std::cout << "CUDA Devices: " << device_count << std::endl;

    // Configure the GPU/CPU selection
    if (mr.flags.no_gpu==0){
	int device;
	if (mr.flags.set_device==1){
	    log(mr.flags.verbose,"CUDA device %d requested.\n",mr.flags.device_number);
	    log(mr.flags.verbose,"Attempting to set device... ");
	    hipSetDevice(mr.flags.device_number);
	    hipGetDevice(&device);
	    if (device!=mr.flags.device_number){
		printf("There was a problem setting device.\n");
	    }
	    else{
		log(mr.flags.verbose,"success!\n");
	    }
	}
	else{
	    hipSetDevice(device_count-1);
	    hipGetDevice(&device);
	}	
	log(mr.flags.verbose,"Working on GPU %i \n",device);
	hipDeviceReset();
    }
    else{
	log(mr.flags.verbose,"Working on CPU\n");
    }

    // --timing cuda events
    TIMER_INIT();    
    TIMER_MASTER_INIT();
    TIMER_MASTER_START();

    /* --- Step 1-3 handled by functions in setup.cu --- */
    // Step 1: Parse input file
    log(mr.flags.verbose,"Reading PRM file...\n");
    if (!exists(argv[argc-1])){
        perror("Parameter file not found. Check path and try again. Exiting:");
        exit(13);            
    }    
    mr.rp=configure_recon_params(argv[argc-1]);

    

    /* --- Check for defined output directory, set to desktop if empty --- */
    /* Configure various file paths and test that we can write out */
    int ctbb_err=configure_paths(&mr);
    log(mr.flags.verbose,"\n");
    log(mr.flags.verbose,"Raw data file:              %s/%s\n",mr.rp.raw_data_dir,mr.rp.raw_data_file);
    log(mr.flags.verbose,"Output file:                %s/%s\n",mr.rp.output_dir,mr.rp.output_file);
    log(mr.flags.verbose,"Testing files written to:   %s/\n",mr.rp.output_dir);
    log(mr.flags.verbose,"Current working directory:  %s/\n",mr.cwd);
    log(mr.flags.verbose,"CTBB exe directory:         %s/\n",mr.install_dir);
    log(mr.flags.verbose,"\n");

    if (ctbb_err){
	switch (ctbb_err){
	case 1:{
	    perror("Cannot read specified raw file.  Ensure the path is correct and you have permission to read the file");
	    break;
	}
	case 2:{
	    perror("Cannot write to specified output file/directory.  Ensure the path is correct and you have permission to write to the directory");
	    break;
	}	    
	}
	exit(13);
    }
    
    // Step 2a: Setup scanner geometry
    log(mr.flags.verbose,"Configuring scanner geometry...\n");
    mr.cg=configure_ct_geom(&mr);
    
    // Step 2b: Configure all remaining information
    
    /* --- Load the dataset into memory --- */
    // Note, we read the whole dataset despite a performance hit.
    // We will work on doing this better in the future.
    std::string raw_data_path = mr.rp.raw_data_dir;
    //std::unique_ptr<fct::RawDataSet> ds = std::make_unique<fct::DicomDataSet>();
    std::shared_ptr<fct::RawDataSet> ds = std::make_shared<fct::DicomDataSet>();
    ds->setPath(raw_data_path);
    ds->initialize();
    ds->readAll();

    log(mr.flags.verbose,"Configuring final reconstruction parameters...\n");
    configure_reconstruction(&mr,ds);

    log(mr.flags.verbose,"Allowed recon range: %.2f to %.2f\n",mr.ri.allowed_begin,mr.ri.allowed_end);
    log(mr.flags.verbose,"\nSTARTING RECONSTRUCTION\n\n");
    
    for (int i=0;i<mr.ri.n_blocks;i++){
	
	update_block_info(&mr);
	
	log(mr.flags.verbose,"----------------------------\n"
	    "Working on block %d of %d \n",i+1,mr.ri.n_blocks);
	
	// Step 3: Extract raw data from file into memory
	log(mr.flags.verbose,"Reading raw data from file...\n");
	extract_projections(&mr,ds);
    
	/* --- Step 3.5: Adaptive filtration handled by preprocessing.cu ---*/
	// Step 3.5: Adaptive filtration of raw data to reduce streak artifacts
	log(mr.flags.verbose,"Running adaptive filtering...\n");

	TIME_EXEC(adaptive_filter_kk(&mr),mr.flags.timing,"adaptive_filtration");

	/* --- Step 4 handled by functions in rebin_filter.cu --- */
	// Step 4: Rebin and filter
	log(mr.flags.verbose,"Rebinning and filtering data...\n");

	if (mr.flags.no_gpu==1){
	    TIME_EXEC(rebin_filter_cpu(&mr),mr.flags.timing,"rebinning and filtering");
	}
	else{
	    TIME_EXEC(rebin_filter(&mr),mr.flags.timing,"rebinning and filtering");
	}

	/* --- Step 5 handled by functions in backproject.cu ---*/
	// Step 5: Backproject
	log(mr.flags.verbose,"Backprojecting...\n");

	if (mr.flags.no_gpu==1){
	    TIME_EXEC(backproject_cpu(&mr),mr.flags.timing,"backprojection");
	}
	else{
	    TIME_EXEC(backproject(&mr),mr.flags.timing,"backprojections");;
	}
    }

    // Step 6: Reorder and thicken slices as needed
    log(mr.flags.verbose,"----------------------------\n");
    log(mr.flags.verbose,"Finalizing image stack...\n");
    
    if (mr.flags.no_gpu==1){
	TIME_EXEC(finalize_image_stack_cpu(&mr),mr.flags.timing,"reordering and thickening slices");
    }
    else{
	TIME_EXEC(finalize_image_stack(&mr),mr.flags.timing,"reordering and thickening slices");
    }

    // Step 7: Save image data to disk (found in setup.cu)
    log(mr.flags.verbose,"----------------------------\n\n");
    log(mr.flags.verbose,"Writing image data to %s/%s\n",mr.rp.output_dir,mr.rp.output_file);
    finish_and_cleanup(&mr);

    TIMER_MASTER_END();
    TIMER_MASTER_PRINT(mr.flags.timing,"entire reconstruction");

    log(mr.flags.verbose,"\nDone.\n");

    hipDeviceReset();
    return 0;
   
}

void log(int verbosity, const char *string,...){
    va_list args;
    va_start(args,string);

    if (verbosity){
	vprintf(string,args);
	va_end(args);
    } 
}
